#include <cucheb.h>

/* driver */
int main(){

  // set device
  hipSetDevice(1);

  // matrix files root directory
  const string rootdir("../matrices/");

  // number of matrices
  const int nummats = 5;

  // lower bounds
  double lb[nummats] = { -0.650,
                         -0.645,
                         -0.640,
                         -0.660,
                         -0.640 };

  // upper bounds
  double ub[nummats] = { -0.00960,
                         -0.00530,
                         -0.00282,
                         -0.33000,
                         -0.00000 };

  // number of trials
  const int numtrials = 3;

  // matrix names
  string matnames[nummats*numtrials] = { "Ge87H76",
                                         "Ge87H76",
                                         "Ge87H76",
                                         "Ge99H100",
                                         "Ge99H100",
                                         "Ge99H100",
                                         "Si41Ge41H72",
                                         "Si41Ge41H72",
                                         "Si41Ge41H72",
                                         "Si87H76",
                                         "Si87H76",
                                         "Si87H76",
                                         "Ga41As41H72",
                                         "Ga41As41H72",
                                         "Ga41As41H72" };

  // matrix names
  int degrees[nummats][numtrials] = { {50,100,-1},
                                      {50,100,-1},
                                      {50,100,-1},
                                      {50,100,-1},
                                      {300,400,-1} };


  // output file
  string ofile("./numex/parsec_data3.txt" );

  // cuchebstats array
  cuchebstats ccstats[nummats*numtrials]; 
  
  // local variables
  string mtxfile;
  cuchebmatrix ccm;
  cucheblanczos ccl;

  // loop through matrices
  for (int ii=0; ii<nummats; ii++) {

    // print matname
    printf(" %s",matnames[ii*numtrials].c_str());

    // set mtxfile
    mtxfile = rootdir + matnames[ii*numtrials] + ".mtx";

    // initialize matrix
    cuchebmatrix_init(mtxfile, &ccm);

    // trials with various degrees
    for (int jj=0; jj<numtrials; jj++) {

      // call filtered lanczos for an interval
      cuchebmatrix_expertlanczos(lb[ii], ub[ii], degrees[ii][jj],
                                 3, DEF_NUM_VECS, DEF_STEP_SIZE,
                                 &ccm, &ccl, &ccstats[ii*numtrials+jj]);

      // print stats
      cuchebstats_print(&ccstats[ii*numtrials+jj]);

      // destroy CCL
      cucheblanczos_destroy(&ccl);

    }

    // destroy CCM
    cuchebmatrix_destroy(&ccm);

  }

  // print ccstats to file
  cuchebstats_fileprint(ofile,nummats*numtrials,&matnames[0],&ccstats[0]);

  // return 
  return 0;

}
