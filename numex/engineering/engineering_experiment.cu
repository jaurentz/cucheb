#include "hip/hip_runtime.h"
#include <cucheb.h>

/* driver */
int main(){

  // set device
  hipSetDevice(0);

  // compute variables
  string temp;
  string rootdir("/home/aurentz/Projects/CUCHEB/cucheb/numex/");
  string matdir("/home/aurentz/Projects/CUCHEB/matrices/");
  ifstream input_file;
  ofstream output_file;
  cuchebmatrix ccm;
  cucheblanczos ccl;
  cuchebstats ccstats;

  // attempt to open output file
  temp = rootdir + "engineering/engineering_data.txt";
  output_file.open( temp.c_str() );
  if (!output_file.is_open()) { 
    printf("Could not open output file.\n");
    exit(1); 
  }

  // variables to parse file
  string matname;
  double a, b, per;
  int deg, bsize, nvecs, ssize;

  // attempt to open input file
  temp = rootdir + "engineering/engineering_matrices.txt";
  input_file.open( temp.c_str() );
  if (!input_file.is_open()) { 
    printf("Could not open matrix file.\n");
    exit(1); 
  }

  // loop through lines
  while (!input_file.eof()) {

    // read in data
    input_file >> matname >> a >> b >> per >> deg >> bsize >> nvecs >> ssize;

    // exit if end of file
    if(input_file.eof()) { break; }

    // initialize matrix
    temp = matdir + matname + ".mtx";
    cuchebmatrix_init(temp, &ccm);

    // call filtered lanczos for an interval
    per = per*abs(b-a);
    cuchebmatrix_expertlanczos(b-per, 1.1*b, deg, bsize, nvecs, ssize,
                                 &ccm, &ccl, &ccstats);

    // print stats
    cuchebstats_print(&ccstats);

  // print eigenvalues
//  for (int ii=0; ii<ccl.nconv; ii++) {
//    printf(" %+e, %e\n",ccl.evals[ccl.index[ii]],ccl.res[ccl.index[ii]]);
//  }
//  printf("\n");

    // write to file
    output_file << matname.c_str() << " "; 
    output_file << b-per << " ";
    output_file << b << " ";
    output_file << ccstats.mat_dim << " ";
    output_file << ccstats.mat_nnz << " ";
    output_file << ccstats.block_size << " ";
    output_file << ccstats.num_blocks << " ";
    output_file << ccstats.num_iters << " ";
    output_file << ccstats.num_innerprods << " ";
    output_file << ccstats.max_degree << " ";
    output_file << ccstats.num_matvecs << " ";
    output_file << ccstats.specint_time << " ";
    output_file << ccstats.innerprod_time << " ";
    output_file << ccstats.matvec_time << " ";
    output_file << ccstats.total_time << " ";
    output_file << ccstats.num_conv << " ";
    output_file << ccstats.max_res << "\n";

    // destroy CCL
    cucheblanczos_destroy(&ccl);

    // read in data
    input_file >> matname >> a >> b >> per >> deg >> bsize >> nvecs >> ssize;

    // exit if end of file
    if(input_file.eof()) { break; }

    // initialize matrix
    temp = matdir + matname + ".mtx";
    cuchebmatrix_init(temp, &ccm);

    // call filtered lanczos for an interval
    per = per*abs(b-a);
    cuchebmatrix_lanczos(b-per, 1.1*b, bsize, nvecs, ssize, &ccm, &ccl, &ccstats);

    // print stats
    cuchebstats_print(&ccstats);

  // print eigenvalues
//  for (int ii=0; ii<ccl.nconv; ii++) {
//    printf(" %+e, %e\n",ccl.evals[ccl.index[ii]],ccl.res[ccl.index[ii]]);
//  }
//  printf("\n");

    // write to file
    output_file << matname.c_str() << " "; 
    output_file << b-per << " ";
    output_file << b << " ";
    output_file << ccstats.mat_dim << " ";
    output_file << ccstats.mat_nnz << " ";
    output_file << ccstats.block_size << " ";
    output_file << ccstats.num_blocks << " ";
    output_file << ccstats.num_iters << " ";
    output_file << ccstats.num_innerprods << " ";
    output_file << ccstats.max_degree << " ";
    output_file << ccstats.num_matvecs << " ";
    output_file << ccstats.specint_time << " ";
    output_file << ccstats.innerprod_time << " ";
    output_file << ccstats.matvec_time << " ";
    output_file << ccstats.total_time << " ";
    output_file << ccstats.num_conv << " ";
    output_file << ccstats.max_res << "\n";

    // destroy cuchebmatrix
    cuchebmatrix_destroy(&ccm);

    // destroy CCL
    cucheblanczos_destroy(&ccl);

  }

  // close input file
  input_file.close();

  // close output file
  output_file.close();

  // return 
  return 0;

}
