#include <cucheb.h>

/* driver */
int main(){

  // set device
  hipSetDevice(1);

  // compute variables
  string temp;
  string rootdir("/home/aurentz/Projects/CUCHEB/cucheb/numex/");
  string matdir("/home/aurentz/Projects/CUCHEB/matrices/");
  ifstream input_file;
  ofstream output_file;
  cuchebmatrix ccm;
  cucheblanczos ccl;
  cuchebstats ccstats;

  // attempt to open output file
  temp = rootdir + "groundstates/groundstates_data.txt";
  output_file.open( temp.c_str() );
  if (!output_file.is_open()) { 
    printf("Could not open output file.\n");
    exit(1); 
  }

  // variables to parse file
  string matname;
  double shift;
  int neigs, deg, bsize, nvecs, ssize;

  // attempt to open input file
  temp = rootdir + "groundstates/groundstates_matrices.txt";
  input_file.open( temp.c_str() );
  if (!input_file.is_open()) { 
    printf("Could not open matrix file.\n");
    exit(1); 
  }

  // loop through lines
  while (!input_file.eof()) {

    // read in data
    input_file >> matname >> shift >> neigs >> deg >> bsize >> nvecs >> ssize;

    // exit if end of file
    if(input_file.eof()) { break; }

    // initialize matrix
    temp = matdir + matname + ".mtx";
    cuchebmatrix_init(temp, &ccm);

    // call filtered lanczos for an interval
    cuchebmatrix_expertlanczos(neigs, shift, deg, bsize, nvecs, ssize,
                                 &ccm, &ccl, &ccstats);

    // print stats
    cuchebstats_print(&ccstats);

    // write to file
    output_file << matname.c_str() << " "; 
    output_file << neigs << " ";
    output_file << ccstats.mat_dim << " ";
    output_file << ccstats.mat_nnz << " ";
    output_file << ccstats.block_size << " ";
    output_file << ccstats.num_blocks << " ";
    output_file << ccstats.num_iters << " ";
    output_file << ccstats.num_innerprods << " ";
    output_file << ccstats.max_degree << " ";
    output_file << ccstats.num_matvecs << " ";
    output_file << ccstats.specint_time << " ";
    output_file << ccstats.arnoldi_time << " ";
    output_file << ccstats.num_conv << " ";
    output_file << ccstats.max_res << "\n";

    // destroy cuchebmatrix
    cuchebmatrix_destroy(&ccm);

    // destroy CCL
    cucheblanczos_destroy(&ccl);

  }

  // close input file
  input_file.close();

  // close output file
  output_file.close();

  // return 
  return 0;

}
