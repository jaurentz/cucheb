#include "hip/hip_runtime.h"
#include <cucheb.h>

/* arnoldi run using cuchebmatrix */
int cucheblanczos_filteredarnoldi(int nsteps, cuchebmatrix* ccm, cuchebpoly* ccp,
                                  cucheblanczos* ccl, cuchebstats* ccstats){

  // local variables
  int n, bsize, nblocks, nvecs, stop;
  double scl, one = 1.0, zero = 0.0, mone = -1.0;
  double* dtemp;
  double* dv1;
  double* dv2;
  double* dschurvecs;
  double* dvecs;
  n = ccl->n;
  bsize = ccl->bsize;
  nblocks = ccl->nblocks;
  nvecs = bsize*nblocks;
  stop = ccl->stop;
  dtemp = ccl->dtemp;
  dschurvecs = ccl->dschurvecs;
  dvecs = ccl->dvecs;
  dv1 = ccl->dv1;
  dv2 = ccl->dv2;
  clock_t tick;

  // set niters
  int niters;
  niters = min(nsteps,nblocks-stop);

  // loop through nblocks
  int ind, odepth, start;
  for(int ii=0; ii < niters; ii++){

    // set index
    ind = (ii+stop)*bsize;

    // time matvecs
    tick = clock();

    // apply filtered matrix
    cuchebmatrix_polymm(ccm,ccp,bsize,&dvecs[ind*n],&dvecs[(ind+bsize)*n],dv1,dv2);
    hipDeviceSynchronize();
    ccstats->matvec_time += (clock()-tick)/((double)CLOCKS_PER_SEC);

    // num_matvecs
    ccstats->num_matvecs += bsize*(ccp->degree);

    // inner loop for bsize blocks
    for(int jj=0; jj < bsize; jj++){

      // time innerprods
      tick = clock();

      // compute orthogonalization depth
      odepth = min((MAX_ORTH_DEPTH)*bsize+jj,ind+bsize);
      start = ind + jj + bsize - odepth;

      // orthogonalize
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, odepth, &one, &dvecs[start*n], 
                  n, &dvecs[(ind+bsize)*n], 1, &zero,
                  &dschurvecs[ind*(nvecs+bsize)+start], 1);
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, odepth, &mone, &dvecs[start*n], 
                  n, &dschurvecs[ind*(nvecs+bsize)+start], 1, &one, &dvecs[(ind+bsize)*n], 1);

      // num_innerprods 
      ccstats->num_innerprods += odepth;

      // reorthogonalize
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, odepth, &one, &dvecs[start*n], 
                  n, &dvecs[(ind+bsize)*n], 1, &zero, &dtemp[0], 1);
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, odepth, &mone, &dvecs[start*n], 
                  n, &dtemp[0], 1, &one, &dvecs[(ind+bsize)*n], 1);
      hipblasDaxpy(ccm->cublashandle, odepth, &one, &dtemp[0], 1, 
                  &dschurvecs[ind*(nvecs+bsize)+start], 1);

      // num_innerprods 
      ccstats->num_innerprods += odepth;

      // normalize
      hipblasDnrm2(ccm->cublashandle, n, &dvecs[(ind+bsize)*n], 1, &scl);
      hipMemcpy(&dschurvecs[ind*(nvecs+bsize)+ind+bsize], &scl,
                 sizeof(double), hipMemcpyHostToDevice);
      scl = 1.0/scl;
      hipblasDscal(ccm->cublashandle, n, &scl, &dvecs[(ind+bsize)*n], 1);
      hipDeviceSynchronize();
      ccstats->innerprod_time += (clock()-tick)/((double)CLOCKS_PER_SEC);

    }

  }

  // update stop 
  ccl->stop += niters;

  // num_blocks
  ccstats->num_blocks += niters;

  // copy data to host
  hipMemcpy(&(ccl->schurvecs)[0],&dschurvecs[0],
             (ccl->stop)*bsize*(nvecs+bsize)*sizeof(double),
             hipMemcpyDeviceToHost);

  // return  
  return 0;

}

