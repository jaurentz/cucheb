#include "hip/hip_runtime.h"
#include <cucheb.h>

/* arnoldi run using cuchebmatrix */
int cucheblanczos_filteredarnoldi(int nsteps, cuchebmatrix* ccm, cuchebpoly* ccp,
                                  cucheblanczos* ccl, cuchebstats* ccstats){

  // local variables
  int n, bsize, nblocks, nvecs, stop;
  double scl, one = 1.0, zero = 0.0, mone = -1.0;
  double* dtemp;
  double* dvecs;
  double* dschurvecs;
  n = ccl->n;
  bsize = ccl->bsize;
  nblocks = ccl->nblocks;
  nvecs = bsize*nblocks;
  stop = ccl->stop;
  dtemp = ccl->dtemp;
  dvecs = ccl->dvecs;
  dschurvecs = ccl->dschurvecs;
  clock_t tick;

  // set niters
  int niters;
  niters = min(nsteps,nblocks-stop);

  // loop through nblocks
  int ind, odepth, start;
  for(int ii=0; ii < niters; ii++){

    // set index
    ind = (ii+stop)*bsize;

    // time matvecs
    tick = clock();

    // apply matrix
///////////////////
NOT DONE HERE
    cuchebmatrix_polymv(ccm,ccp,&dvecs[ind*n],&dvecs[(ind+bsize)*n]);
    hipDeviceSynchronize();
    ccstats->matvec_time += (clock()-tick)/((double)CLOCKS_PER_SEC);
///////////////////

    // num_matvecs
    ccstats->num_matvecs += bsize*(ccp->degree);

    // inner loop for bsize blocks
    for(int jj=0; jj < bsize; jj++){

      // time innerprods
      tick = clock();

      // compute orthogonalization depth
      odepth = min((MAX_ORTH_DEPTH)*bsize+jj,ind+bsize);
      start = ind + jj + bsize - odepth;

      // orthogonalize
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, odepth, &one, &dvecs[start*n], 
                  n, &dvecs[(ind+bsize)*n], 1, &zero,
                  &dschurvecs[ind*(nvecs+bsize)+start], 1);
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, odepth, &mone, &dvecs[start*n], 
                  n, &dschurvecs[ind*(nvecs+bsize)+start], 1, &one, &dvecs[(ind+bsize)*n], 1);

      // num_innerprods 
      ccstats->num_innerprods += odepth;

      // reorthogonalize
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, odepth, &one, &dvecs[start*n], 
                  n, &dvecs[(ind+bsize)*n], 1, &zero, &dtemp[0], 1);
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, odepth, &mone, &dvecs[start*n], 
                  n, &dtemp[0], 1, &one, &dvecs[(ind+bsize)*n], 1);
      hipblasDaxpy(ccm->cublashandle, odepth, &one, &dtemp[0], 1, 
                  &dschurvecs[ind*(nvecs+bsize)+start], 1);

      // num_innerprods 
      ccstats->num_innerprods += odepth;

      // normalize
      hipblasDnrm2(ccm->cublashandle, n, &dvecs[(ind+bsize)*n], 1, &scl);
      hipMemcpy(&dschurvecs[ind*(nvecs+bsize)+ind+bsize], &scl,
                 sizeof(double), hipMemcpyHostToDevice);
      scl = 1.0/scl;
      hipblasDscal(ccm->cublashandle, n, &scl, &dvecs[(ind+bsize)*n], 1);
      hipDeviceSynchronize();
      ccstats->innerprod_time += (clock()-tick)/((double)CLOCKS_PER_SEC);

    }

  }

  // update stop 
  ccl->stop += niters;

  // num_blocks
  ccstats->num_blocks += niters;

  // copy data to host
  hipMemcpy(&(ccl->schurvecs)[0],&dschurvecs[0],
             (ccl->stop)*bsize*(nvecs+bsize)*sizeof(double),
             hipMemcpyDeviceToHost);

  // return  
  return 0;

}

