#include "hip/hip_runtime.h"
#include <cucheb.h>

/* arnoldi run using cuchebmatrix */
int cucheblanczos_filteredarnoldi(cuchebmatrix* ccm, cuchebpoly* ccp,
                                  cucheblanczos* ccl, cuchebstats* ccstats){

  // local variables
  int n, bsize, nblocks, nvecs, odepth;
  double scl, one = 1.0, zero = 0.0, mone = -1.0;
  double* bands;
  double* dtemp;
  double* dvecs;
  double* dschurvecs;
  n = ccl->n;
  bsize = ccl->bsize;
  nblocks = ccl->nblocks;
  nvecs = bsize*nblocks;
  bands = ccl->bands;
  dtemp = ccl->dtemp;
  dvecs = ccl->dvecs;
  dschurvecs = ccl->dschurvecs;

  // loop through nblocks
  int ind, start;
  for(int ii=0; ii < nblocks; ii++){

    // inner loop for bsize blocks
    for(int jj=0; jj < bsize; jj++){

      // set index
      ind = ii*bsize + jj;

      // apply matrix
      cuchebmatrix_polymv(ccm,ccp,&dvecs[ind*n],&dvecs[(ind+bsize)*n]);

      // num_matvecs
      ccstats->num_matvecs += (ccp->degree);

      // compute orthogonalization depth
      odepth = min((MAX_ORTH_DEPTH)*bsize+jj,ind+bsize);
      start = ind + bsize - odepth;

      // orthogonalize
//      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, (ind+bsize), &one, &dvecs[0], n, 
//                  &dvecs[(ind+bsize)*n], 1, &zero, &dschurvecs[ind*(nvecs+bsize)], 1);
//      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, (ind+bsize), &mone, &dvecs[0], n, 
//                  &dschurvecs[ind*(nvecs+bsize)], 1, &one, &dvecs[(ind+bsize)*n], 1);

      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, odepth, &one, &dvecs[start*n], 
                  n, &dvecs[(ind+bsize)*n], 1, &zero, &dschurvecs[ind*(nvecs+bsize)+start], 1);
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, odepth, &mone, &dvecs[start*n], 
                  n, &dschurvecs[ind*(nvecs+bsize)+start], 1, &one, &dvecs[(ind+bsize)*n], 1);

      // num_innerprods 
      ccstats->num_innerprods += odepth;

      // reorthogonalize
//      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, (ind+bsize), &one, &dvecs[0], n, 
//                &dvecs[(ind+bsize)*n], 1, &zero, &dtemp[0], 1);
//      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, (ind+bsize), &mone, &dvecs[0], n, 
//                &dtemp[0], 1, &one, &dvecs[(ind+bsize)*n], 1);
//      hipblasDaxpy(ccm->cublashandle, (ind+bsize), &one, &dtemp[0], 1, 
//                &dschurvecs[ind*(nvecs+bsize)], 1);

      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_T, n, odepth, &one, &dvecs[start*n], 
                  n, &dvecs[(ind+bsize)*n], 1, &zero, &dtemp[0], 1);
      hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, odepth, &mone, &dvecs[start*n], 
                  n, &dtemp[0], 1, &one, &dvecs[(ind+bsize)*n], 1);
      hipblasDaxpy(ccm->cublashandle, odepth, &one, &dtemp[0], 1, 
                  &dschurvecs[ind*(nvecs+bsize)+start], 1);

      // num_innerprods 
      ccstats->num_innerprods += odepth;

      // normalize
      hipblasDnrm2(ccm->cublashandle, n, &dvecs[(ind+bsize)*n], 1,
                  &bands[(ind+1)*(bsize+1)-1]);
      scl = 1.0/bands[(ind+1)*(bsize+1)-1];
      hipblasDscal(ccm->cublashandle, n, &scl, &dvecs[(ind+bsize)*n], 1);

    }

  }

  // copy data to host
  hipMemcpy(&(ccl->schurvecs)[0],&dschurvecs[0],nvecs*(nvecs+bsize)*sizeof(double),
             hipMemcpyDeviceToHost);

  // return  
  return 0;

}

