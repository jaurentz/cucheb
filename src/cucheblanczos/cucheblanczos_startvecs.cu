#include "hip/hip_runtime.h"
#include <cucheb.h>

/* routine to create starting vector for cucheblanczos */
int cucheblanczos_startvecs(cucheblanczos* ccl){

  // initial starting vector to be normalized all ones vector
  double scl;
  scl = 1.0/sqrt(1.0*ccl->n);
  for(int ii=0; ii < ccl->n; ii++){
    hipMemcpy(&(ccl->dvecs)[ii],&scl,sizeof(double),hipMemcpyHostToDevice);
  }

  // all other starting vectors
  double one = 1.0/sqrt(2.0), mone = -1.0/sqrt(2.0), zero = 0.0;
  for(int jj=1; jj < ccl->bsize; jj++){
    for(int ii=0; ii < ccl->n; ii++){
      if (ii == 2*(jj-1)) {
        hipMemcpy(&(ccl->dvecs)[jj*(ccl->n) + ii],&one,sizeof(double),hipMemcpyHostToDevice);
      }
      else if (ii == 2*(jj-1)+1) {
        hipMemcpy(&(ccl->dvecs)[jj*(ccl->n) + ii],&mone,sizeof(double),hipMemcpyHostToDevice);
      }
      else {
        hipMemcpy(&(ccl->dvecs)[jj*(ccl->n) + ii],&zero,sizeof(double),hipMemcpyHostToDevice);
      }
    }
  }

  // return  
  return 0;

}
