#include "hip/hip_runtime.h"
#include <cucheb.h>

/* compute ritz values */
int cucheblanczos_ritz(cuchebmatrix* ccm, cucheblanczos* ccl){

  // local variables
  int bsize, nvecs, stop;
  double V[MAX_BLOCK_SIZE] = {0.0};
  double R[(MAX_BLOCK_SIZE)*(MAX_BLOCK_SIZE)] = {0.0};
  double* evals;
  double* res;
  double* bands;
  double* schurvecs;
  bsize = ccl->bsize;
  nvecs = (ccl->bsize)*(ccl->nblocks);
  stop = ccl->stop;
  evals = ccl->evals;
  res = ccl->res;
  bands = ccl->bands;
  schurvecs = ccl->schurvecs;

  // fill bands
  for(int ii=0; ii<bsize+1; ii++){
    for(int jj=0; jj<stop*bsize; jj++) {
      bands[jj*(bsize+1)+ii] = schurvecs[jj*(nvecs+bsize)+jj+ii];
    }
  }

  // fill R
  int ind;
  for(int ii=0; ii < bsize; ii++){
    ind = ((stop-1)*bsize+ii)*(nvecs+bsize+1)+bsize;
    for(int jj=0; jj < ii+1; jj++){
      R[ii*(MAX_BLOCK_SIZE + 1)-jj] = schurvecs[ind-jj];
    }
  }
  
  // initialize schurvectors
  for(int ii=0; ii<nvecs+bsize; ii++){
    for(int jj=0; jj<stop*bsize; jj++) {
      if (ii == jj){ schurvecs[jj*(nvecs+bsize)+ii] = 1.0; }
      else{ schurvecs[jj*(nvecs+bsize)+ii] = 0.0; }
    }
  }

  // call bandsymqr
  cuchebutils_bandsymqr(stop*bsize, bsize+1, bands, bsize+1,
                evals, schurvecs, nvecs+bsize);

  // compute residuals
  double tmp = 0;
  for(int ii=0; ii < stop*bsize; ii++){

    // set res[ii] to 0
    res[ii] = 0.0;

    // set V
    for(int jj=0; jj<bsize; jj++){
      V[jj] = schurvecs[ii*(nvecs+bsize) + (stop-1)*bsize + jj];
    }
  
    // compute matrix vector product and norm 
    for(int jj=0; jj<bsize; jj++){
      tmp = 0.0;
      for(int kk=0; kk<bsize-jj; kk++){
        tmp += R[kk*MAX_BLOCK_SIZE+jj]*V[kk];
      }
      res[ii] += tmp*tmp;
    }

    // compute sqrt
    res[ii] = sqrt(res[ii]);

  }

  // set nconv
  ccl->nconv = stop*bsize;

  // return  
  return 0;

}
