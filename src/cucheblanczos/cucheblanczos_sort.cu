#include "hip/hip_runtime.h"
#include <cucheb.h>

/* my interval norm */
double interval_norm(double lb, double ub, double val){

  if (val >= lb && val <= ub) { return val; }
  else { return abs(val) + ub; }

}

/* sort evals in interval */
int cucheblanczos_sort(double lb, double ub, cucheblanczos* ccl){

  // local variables
  int neig;
  int* index;
  double* evals;
  neig = (ccl->nconv);
  index = ccl->index;
  evals = ccl->evals;

  // sort ritz values
  // create a vector of evals and indices
  vector< pair< double , int > > temp;
  for(int ii=0; ii < neig; ii++){
    temp.push_back(make_pair( interval_norm(lb,ub,evals[index[ii]]), index[ii] ));
  }

  // sort vector
  sort(temp.begin(),temp.end());
 
  // update index
  for(int ii=0; ii < neig; ii++){
    index[ii] = temp[ii].second;
  }

  // compute number of converged eigenvalues
  ccl->nconv = 0;
  for(int ii=0; ii < neig; ii++){
    if (evals[index[ii]] > ub || evals[index[ii]] < lb){ break; }
    ccl->nconv += 1;  
  }

  // return  
  return 0;

}



/* sort evals by largest modulus*/
int cucheblanczos_sort(cucheblanczos* ccl){

  // local variables
  int neig;
  int* index;
  double* evals;
  neig = (ccl->nconv);
  index = ccl->index;
  evals = ccl->evals;

  // sort ritz values
  // create a vector of evals and indices
  vector< pair< double , int > > temp;
  for(int ii=0; ii < neig; ii++){
    temp.push_back(make_pair( -evals[index[ii]], index[ii] ));
  }

  // sort vector
  sort(temp.begin(),temp.end());

  // update index
  for(int ii=0; ii < neig; ii++){
    index[ii] = temp[ii].second;
  }

  // return  
  return 0;

}
