#include "hip/hip_runtime.h"
#include <cucheb.h>

/* compute ritz values and vectors */
int cucheblanczos_rayleigh(cuchebmatrix* ccm, cucheblanczos* ccl){

  // local variables
  int n, bsize, nvecs, stop;
  int* index;
  double* evals;
  double* res;
  double* vecs;
  double* schurvecs;
  double* dv1;
  double* dv2;
  double* dschurvecs;
  double* dvecs;
  n = ccl->n;
  bsize = ccl->bsize;
  stop = ccl->stop;
  nvecs = bsize*(ccl->nblocks);
  index = ccl->index;
  evals = ccl->evals;
  res = ccl->res;
  schurvecs = ccl->schurvecs;
  vecs = ccl->vecs;
  dv1 = &(ccm->dtemp)[0];
  dv2 = &(ccm->dtemp)[n];
  dschurvecs = ccl->dschurvecs;
  dvecs = ccl->dvecs;

  // copy schurvecs into dschur
  for(int ii=0; ii<stop*bsize; ii++){
    hipMemcpy(&dschurvecs[ii*(nvecs+bsize)],&schurvecs[index[ii]*(nvecs+bsize)],
               (nvecs+bsize)*sizeof(double),hipMemcpyHostToDevice);
  }

  // compute rayleigh quotients and residuals
  double one = 1.0, zero = 0.0;
  double scl, rval;
  for(int ii=0; ii<stop*bsize; ii++){
 
    // compute ritz vector
    hipblasDgemv(ccm->cublashandle, HIPBLAS_OP_N, n, stop*bsize, &one, dvecs, 
                n, &dschurvecs[ii*(nvecs+bsize)], 1, &zero, dv1, 1);

    // copy ritz vector to cpu
    hipMemcpy(&vecs[ii*n],dv1,n*sizeof(double),hipMemcpyDeviceToHost);

    // apply operator
    cuchebmatrix_mv(ccm,&one,dv1,&zero,dv2);

    // compute rayleigh quotient
    hipblasDnrm2(ccm->cublashandle,n,dv1,1,&scl);
    hipblasDdot(ccm->cublashandle,n,dv1,1,dv2,1,&evals[ii]);
    evals[ii] = evals[ii]/scl/scl;

    // compute residual vector
    rval = -evals[ii];
    hipblasDaxpy(ccm->cublashandle,n,&rval,dv1,1,dv2,1);

    // compute norm of residual
    hipblasDnrm2(ccm->cublashandle,n,dv2,1,&res[ii]);
    res[ii] = res[ii]/scl;

    // reset index 
    index[ii] = ii;
  
  }

  // return  
  return 0;

}
