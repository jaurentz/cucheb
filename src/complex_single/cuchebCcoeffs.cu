#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------



-----------------------------------------------------------------*/

#include <cucheb.h>


/* chebcoeffs */
/* complex single precision */
__global__ void cinput (int n, const hipComplex *fvals, int incfvals, hipfftComplex *input){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;
	int N = 2*(n-1);

	if(ii == 0){
		input[ii] = fvals[(n-1)*incfvals];
	}
	else if(ii == n-1){
		input[ii] = fvals[0];
	}
	else if(ii > 0 && ii < n-1){
		input[ii] = fvals[(n-1-ii)*incfvals];
		input[N-ii] = fvals[(n-1-ii)*incfvals];
	}
}
__global__ void coutput (int n, const hipfftComplex *output, hipComplex *coeffs, int inccfs){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;
	hipComplex scl;

	if(ii == 0){
		scl = make_hipFloatComplex((float)(n-1)*2.0f,0.0f);
		coeffs[ii*inccfs] = hipCdivf(output[n-1-ii],scl);
	}
	else if(ii == n-1){
		scl = make_hipFloatComplex((float)(n-1)*2.0f,0.0f);
		coeffs[ii*inccfs] = hipCdivf(output[n-1-ii],scl);
	}
	else if(ii > 0 && ii < n-1){
		scl = make_hipFloatComplex((float)(n-1),0.0f);
		coeffs[ii*inccfs] = hipCdivf(output[n-1-ii],scl);
	}
}
cuchebStatus_t cuchebCcoeffs (int n, const hipComplex *fvals, int incfvals, hipComplex *coeffs, int inccfs){

	// check n
	if(n <= 0){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	else if(n > MAX_FLOAT_DEG+1){
		fprintf(stderr,"\nIn %s line: %d, n must be <= %d.\n",__FILE__,__LINE__,MAX_FLOAT_DEG+1);
		cuchebExit(-1);
	}
	
	// check incfvals
	if(incfvals <= 0){
		fprintf(stderr,"\nIn %s line: %d, incfvals must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check inccfs
	if(inccfs <= 0){
		fprintf(stderr,"\nIn %s line: %d, inccfs must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// n = 1
	if(n == 1){
		cuchebCheckError(hipMemcpy(coeffs, fvals, sizeof(hipComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	
	// n > 1
	else{
		// allocate workspace
		hipfftComplex *input;
		cuchebCheckError(hipMalloc(&input, 2*(n-1)*sizeof(hipfftComplex)),__FILE__,__LINE__);
	
		// query device
		int dev;
		hipDeviceProp_t prop;
		cuchebCheckError(hipGetDevice(&dev),__FILE__,__LINE__);
		cuchebCheckError(hipGetDeviceProperties(&prop,dev),__FILE__,__LINE__);
	
		// set blockSize
		int blockSize;
		blockSize = prop.maxThreadsPerBlock;
	
		// set gridSize
		int gridSize;
		gridSize = (int)ceil((double)n/blockSize);
	
		// launch fill input kernel
		cinput<<<gridSize,blockSize>>>(n,fvals,incfvals,input);
	
		// check for kernel error
		cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
		// initialize cufft
		hipfftHandle cufftHand;
		cuchebCheckError(hipfftPlan1d(&cufftHand, 2*(n-1), HIPFFT_C2C, 1),__FILE__,__LINE__);
	
		// execute plan
		cuchebCheckError(hipfftExecC2C(cufftHand,input,input,HIPFFT_FORWARD),__FILE__,__LINE__);
	
		// launch extract output kernel
		coutput<<<gridSize,blockSize>>>(n,input,coeffs,inccfs);
	
		// check for kernel error
		cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
		// free cufft
		cuchebCheckError(hipfftDestroy(cufftHand),__FILE__,__LINE__);
	
		// free workspace
		cuchebCheckError(hipFree(input),__FILE__,__LINE__);
	}
	
	// return success
	return CUCHEB_STATUS_SUCCESS;
}
