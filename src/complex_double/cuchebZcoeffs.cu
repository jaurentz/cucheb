#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------



-----------------------------------------------------------------*/

#include <cucheb.h>


/* chebcoeffs */
/* complex double precision */
__global__ void zinput (int n, const hipDoubleComplex *fvals, int incfvals, hipfftDoubleComplex *input){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;
	int N = 2*(n-1);

	if(ii == 0){
		input[ii] = fvals[(n-1)*incfvals];
	}
	else if(ii == n-1){
		input[ii] = fvals[0];
	}
	else if(ii > 0 && ii < n-1){
		input[ii] = fvals[(n-1-ii)*incfvals];
		input[N-ii] = fvals[(n-1-ii)*incfvals];
	}
}
__global__ void zoutput (int n, const hipfftDoubleComplex *output, hipDoubleComplex *coeffs, int inccfs){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;
	hipDoubleComplex scl;

	if(ii == 0){
		scl = make_hipDoubleComplex((double)(n-1)*2.0,0.0);
		coeffs[ii*inccfs] = hipCdiv(output[n-1-ii],scl);
	}
	else if(ii == n-1){
		scl = make_hipDoubleComplex((double)(n-1)*2.0,0.0);
		coeffs[ii*inccfs] = hipCdiv(output[n-1-ii],scl);
	}
	else if(ii > 0 && ii < n-1){
		scl = make_hipDoubleComplex((double)(n-1),0.0);
		coeffs[ii*inccfs] = hipCdiv(output[n-1-ii],scl);
	}
}
cuchebStatus_t cuchebZcoeffs (int n, const hipDoubleComplex *fvals, int incfvals, hipDoubleComplex *coeffs, int inccfs){

	// check n
	if(n <= 0){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	else if(n > MAX_DOUBLE_DEG+1){
		fprintf(stderr,"\nIn %s line: %d, n must be <= %d.\n",__FILE__,__LINE__,MAX_DOUBLE_DEG+1);
		cuchebExit(-1);
	}
	
	// check incfvals
	if(incfvals <= 0){
		fprintf(stderr,"\nIn %s line: %d, incfvals must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check inccfs
	if(inccfs <= 0){
		fprintf(stderr,"\nIn %s line: %d, inccfs must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// n = 1
	if(n == 1){
		cuchebCheckError(hipMemcpy(coeffs, fvals, sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	
	// n > 1
	else{
		// allocate workspace
		hipfftDoubleComplex *input;
		cuchebCheckError(hipMalloc(&input, 2*(n-1)*sizeof(hipfftDoubleComplex)),__FILE__,__LINE__);
	
		// query device
		int dev;
		hipDeviceProp_t prop;
		cuchebCheckError(hipGetDevice(&dev),__FILE__,__LINE__);
		cuchebCheckError(hipGetDeviceProperties(&prop,dev),__FILE__,__LINE__);
	
		// set blockSize
		int blockSize;
		blockSize = prop.maxThreadsPerBlock;
	
		// set gridSize
		int gridSize;
		gridSize = (int)ceil((double)n/blockSize);
	
		// launch fill input kernel
		zinput<<<gridSize,blockSize>>>(n,fvals,incfvals,input);
	
		// check for kernel error
		cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
		// initialize cufft
		hipfftHandle cufftHand;
		cuchebCheckError(hipfftPlan1d(&cufftHand, 2*(n-1), HIPFFT_Z2Z, 1),__FILE__,__LINE__);
	
		// execute plan
		cuchebCheckError(hipfftExecZ2Z(cufftHand,input,input,HIPFFT_FORWARD),__FILE__,__LINE__);
	
		// launch extract output kernel
		zoutput<<<gridSize,blockSize>>>(n,input,coeffs,inccfs);
	
		// check for kernel error
		cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
		// free cufft
		cuchebCheckError(hipfftDestroy(cufftHand),__FILE__,__LINE__);
	
		// free workspace
		cuchebCheckError(hipFree(input),__FILE__,__LINE__);
	}
	
	// return success
	return CUCHEB_STATUS_SUCCESS;
}
