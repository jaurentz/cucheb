#include "hip/hip_runtime.h"
#include <cucheb.h>

/* complex hipDoubleComplex precision constructors */
/* fixed degree */
ChebPoly::ChebPoly(cuchebCuDoubleComplexFun fun, hipDoubleComplex *A, hipDoubleComplex *B, void *USERDATA, int Deg){

	// set field
	field = CUCHEB_FIELD_DOUBLE_COMPLEX;

	// check degree
	if(Deg < 0){
		fprintf(stderr,"\nIn %s line: %d, degree must be >= 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	if(Deg > MAX_DOUBLE_DEG){
		fprintf(stderr,"\nIn %s line: %d, degree must be <= %d.\n",__FILE__,__LINE__,MAX_DOUBLE_DEG);
		cuchebExit(-1);
	}
	
	// set degree
	degree = Deg;
	
	// check a and b
	if(hipCreal(*A) == hipCreal(*B) && hipCimag(*A) == hipCimag(*B)){
		fprintf(stderr,"\nIn %s line: %d, a must not = b.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// set a and b
	cuchebCheckError(hipMalloc(&a, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&b, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(a, A, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(b, B, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// degree 0
	if(degree == 0){
		// compute funvals
		hipDoubleComplex *sfvs;
		cuchebCheckError(hipMalloc(&sfvs, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError((*fun)(1, (hipDoubleComplex*)a, 1, sfvs, 1, USERDATA),__FILE__,__LINE__);
		
		// set coeffs
		cuchebCheckError(hipMalloc(&coeffs, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(coeffs, sfvs, sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);

		// free device memory
		cuchebCheckError(hipFree(sfvs),__FILE__,__LINE__);	
	}
	
	// degree > 0
	else{
		// compute chebpoints
		hipDoubleComplex *spts;
		cuchebCheckError(hipMalloc(&spts, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError(cuchebZpoints(degree+1, (hipDoubleComplex*)a, (hipDoubleComplex*)b, spts, 1),__FILE__,__LINE__);
	
		// compute funvals
		hipDoubleComplex *sfvs;
		cuchebCheckError(hipMalloc(&sfvs, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError((*fun)(degree+1, spts, 1, sfvs, 1, USERDATA),__FILE__,__LINE__);
		
		// compute chebcoeffs
		hipDoubleComplex *scfs;
		cuchebCheckError(hipMalloc(&scfs, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError(cuchebZcoeffs(degree+1, sfvs, 1, scfs, 1),__FILE__,__LINE__);
		
		// set coeffs
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(coeffs, scfs, (degree+1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);

		// free device memory
		cuchebCheckError(hipFree(spts),__FILE__,__LINE__);
		cuchebCheckError(hipFree(sfvs),__FILE__,__LINE__);
		cuchebCheckError(hipFree(scfs),__FILE__,__LINE__);
	}
}

/* user specified tolerance */
ChebPoly::ChebPoly(cuchebCuDoubleComplexFun fun, hipDoubleComplex *A, hipDoubleComplex *B, void *USERDATA, double *tol){

	// set field
	field = CUCHEB_FIELD_DOUBLE_COMPLEX;
	
	// check a and b
	if(hipCreal(*A) == hipCreal(*B) && hipCimag(*A) == hipCimag(*B)){
		fprintf(stderr,"\nIn %s line: %d, a must not = b.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// set a and b
	cuchebCheckError(hipMalloc(&a, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&b, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(a, A, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(b, B, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// check tol
	if(*tol <= 0){
		fprintf(stderr,"\nIn %s line: %d, tol must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// compute chebpoints
	hipDoubleComplex *zpts;
	cuchebCheckError(hipMalloc(&zpts, (MAX_DOUBLE_DEG+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(cuchebZpoints(MAX_DOUBLE_DEG+1, (hipDoubleComplex*)a, (hipDoubleComplex*)b, zpts, 1),__FILE__,__LINE__);
	
	// compute funvals
	hipDoubleComplex *zfvs;
	cuchebCheckError(hipMalloc(&zfvs, (MAX_DOUBLE_DEG+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError((*fun)(MAX_DOUBLE_DEG+1, zpts, 1, zfvs, 1, USERDATA),__FILE__,__LINE__);
		
	/* compute chebcoeffs */
	// initialize zcfs
	hipDoubleComplex *zcfs;
	cuchebCheckError(hipMalloc(&zcfs, (MAX_DOUBLE_DEG+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	
	// initialize compute variables
	int stride = pow(2,MAX_DOUBLE_DEG_EXP-3); 
	int current_degree = pow(2,3);
	int max_index;
	int start_index = 0;
	bool converged = false;
	double max_abs, current_abs;
	hipDoubleComplex *max_val, *current_val;
	
	// allocate host pointers
	cuchebCheckError((void*)(max_val = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex))),__FILE__,__LINE__);
	cuchebCheckError((void*)(current_val = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex))),__FILE__,__LINE__);
	
	// initialize cublas
	hipblasHandle_t cublasHand;
	cuchebCheckError(hipblasCreate(&cublasHand),__FILE__,__LINE__);
	cuchebCheckError(hipblasSetPointerMode(cublasHand, HIPBLAS_POINTER_MODE_HOST),__FILE__,__LINE__);
	
	// compute coeffs adaptively until convergence
	while(converged != true){
		// compute cheb interpolant of current_degree 
		cuchebCheckError(cuchebZcoeffs(current_degree+1, zfvs, stride, zcfs, 1),__FILE__,__LINE__);

		// get max_index
		cuchebCheckError(hipblasIzamax(cublasHand, current_degree+1, zcfs, 1, &max_index),__FILE__,__LINE__);
		
		// set maximum modulus of coefficient
		cuchebCheckError(hipMemcpy(max_val, &zcfs[max_index-1], sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		max_abs = hipCabs(*max_val);

		// check for convergence
		for(int ii=0;ii<current_degree;ii++){
			// get current coefficient
			cuchebCheckError(hipMemcpy(current_val, &zcfs[ii], sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
			current_abs = hipCabs(*current_val);
			
			// check first coeff
			if(current_abs >= (*tol)*max_abs && ii == 0){
				stride = stride/2;
				current_degree = current_degree*2;
				converged = false;
				break;
			}
			// check second coeff
			else if(current_abs >= (*tol)*max_abs && ii == 1){
				stride = stride/2;
				current_degree = current_degree*2;
				converged = false;
				break;
			}
			// check middle coeffs
			else if(current_abs >= (*tol)*max_abs && ii > 1){
				degree = current_degree-ii;
				start_index = ii;
				converged = true;
				break;
			}
			// last coeff
			else if(ii == current_degree-1){
				degree = 0;
				start_index = current_degree;
				converged = true;
				break;
			}
		}
		
		// check current_degree
		if(current_degree > MAX_DOUBLE_DEG){
			printf("\nWarning in %s line: %d\n Function could not be resolved to specified tolerance %e, by a %d degree ChebPoly!\n\n",
				__FILE__,__LINE__,*tol,MAX_DOUBLE_DEG);

			degree = MAX_DOUBLE_DEG;
			start_index = 0;
			converged = true;
		}
	}
	// free host pointers
	free(max_val);
	free(current_val);
	
	// free cublas
	cuchebCheckError(hipblasDestroy(cublasHand),__FILE__,__LINE__);
	/* end compute chebcoeffs */
		
	// set coeffs
	cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(coeffs, &zcfs[start_index], (degree+1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);

	// free device memory
	cuchebCheckError(hipFree(zpts),__FILE__,__LINE__);
	cuchebCheckError(hipFree(zfvs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(zcfs),__FILE__,__LINE__);
}

/* default tolerance */
ChebPoly::ChebPoly(cuchebCuDoubleComplexFun fun, hipDoubleComplex *A, hipDoubleComplex *B, void *USERDATA){

	// set field
	field = CUCHEB_FIELD_DOUBLE_COMPLEX;
	
	// check a and b
	if(hipCreal(*A) == hipCreal(*B) && hipCimag(*A) == hipCimag(*B)){
		fprintf(stderr,"\nIn %s line: %d, a must not = b.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// set a and b
	cuchebCheckError(hipMalloc(&a, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&b, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(a, A, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(b, B, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// compute chebpoints
	hipDoubleComplex *zpts;
	cuchebCheckError(hipMalloc(&zpts, (MAX_DOUBLE_DEG+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(cuchebZpoints(MAX_DOUBLE_DEG+1, (hipDoubleComplex*)a, (hipDoubleComplex*)b, zpts, 1),__FILE__,__LINE__);
	
	// compute funvals
	hipDoubleComplex *zfvs;
	cuchebCheckError(hipMalloc(&zfvs, (MAX_DOUBLE_DEG+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError((*fun)(MAX_DOUBLE_DEG+1, zpts, 1, zfvs, 1, USERDATA),__FILE__,__LINE__);
		
	/* compute chebcoeffs */
	// initialize zcfs
	hipDoubleComplex *zcfs;
	cuchebCheckError(hipMalloc(&zcfs, (MAX_DOUBLE_DEG+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	
	// initialize compute variables
	int stride = pow(2,MAX_DOUBLE_DEG_EXP-3); 
	int current_degree = pow(2,3);
	int max_index;
	int start_index = 0;
	bool converged = false;
	double max_abs, current_abs;
	hipDoubleComplex *max_val, *current_val;
	
	// allocate host pointers
	cuchebCheckError((void*)(max_val = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex))),__FILE__,__LINE__);
	cuchebCheckError((void*)(current_val = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex))),__FILE__,__LINE__);
	
	// initialize cublas
	hipblasHandle_t cublasHand;
	cuchebCheckError(hipblasCreate(&cublasHand),__FILE__,__LINE__);
	cuchebCheckError(hipblasSetPointerMode(cublasHand, HIPBLAS_POINTER_MODE_HOST),__FILE__,__LINE__);
	
	// compute coeffs adaptively until convergence
	while(converged != true){
		// compute cheb interpolant of current_degree 
		cuchebCheckError(cuchebZcoeffs(current_degree+1, zfvs, stride, zcfs, 1),__FILE__,__LINE__);

		// get max_index
		cuchebCheckError(hipblasIzamax(cublasHand, current_degree+1, zcfs, 1, &max_index),__FILE__,__LINE__);
		
		// set maximum modulus of coefficient
		cuchebCheckError(hipMemcpy(max_val, &zcfs[max_index-1], sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		max_abs = hipCabs(*max_val);

		// check for convergence
		for(int ii=0;ii<current_degree;ii++){
			// get current coefficient
			cuchebCheckError(hipMemcpy(current_val, &zcfs[ii], sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
			current_abs = hipCabs(*current_val);
			
			// check first coeff
			if(current_abs >= DBL_EPSILON*max_abs && ii == 0){
				stride = stride/2;
				current_degree = current_degree*2;
				converged = false;
				break;
			}
			// check second coeff
			else if(current_abs >= DBL_EPSILON*max_abs && ii == 1){
				stride = stride/2;
				current_degree = current_degree*2;
				converged = false;
				break;
			}
			// check middle coeffs
			else if(current_abs >= DBL_EPSILON*max_abs && ii > 1){
				degree = current_degree-ii;
				start_index = ii;
				converged = true;
				break;
			}
			// last coeff
			else if(ii == current_degree-1){
				degree = 0;
				start_index = current_degree;
				converged = true;
				break;
			}
		}
		
		// check current_degree
		if(current_degree > MAX_DOUBLE_DEG){
			printf("\nWarning in %s line: %d\n Function could not be resolved to machine tolerance %e, by a %d degree ChebPoly!\n\n",
				__FILE__,__LINE__,DBL_EPSILON,MAX_DOUBLE_DEG);

			degree = MAX_DOUBLE_DEG;
			start_index = 0;
			converged = true;
		}
	}
	// free host pointers
	free(max_val);
	free(current_val);
	
	// free cublas
	cuchebCheckError(hipblasDestroy(cublasHand),__FILE__,__LINE__);
	/* end compute chebcoeffs */
		
	// set coeffs
	cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(coeffs, &zcfs[start_index], (degree+1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);

	// free device memory
	cuchebCheckError(hipFree(zpts),__FILE__,__LINE__);
	cuchebCheckError(hipFree(zfvs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(zcfs),__FILE__,__LINE__);
}
