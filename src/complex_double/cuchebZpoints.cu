#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------



-----------------------------------------------------------------*/

#include <cucheb.h>


/* chebpoints */
/* complex double precision */
__global__ void zpoints (int n,const hipDoubleComplex *a,const hipDoubleComplex *b,hipDoubleComplex *pts,int incpts){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;
	double theta;
	hipDoubleComplex two = make_hipDoubleComplex(2.0,0.0);
	hipDoubleComplex sine;

	if(ii < n){
		theta = (double)(M_PI_2)*(2*ii-n+1)/(n-1);
		sine = make_hipDoubleComplex(sin(theta),0.0);
		pts[ii*incpts] = hipCdiv(hipCadd(hipCadd(*b,*a),hipCmul(sine,hipCsub(*b,*a))),two);
	}
}
cuchebStatus_t cuchebZpoints (int n,const hipDoubleComplex *a,const hipDoubleComplex *b,hipDoubleComplex *pts,int incpts){

	// check n
	if(n <= 1){
		fprintf(stderr,"\nIn %s line: %d, n must be > 1.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	if(n > MAX_DOUBLE_DEG+1){
		fprintf(stderr,"\nIn %s line: %d, n must be <= %d.\n",__FILE__,__LINE__,MAX_DOUBLE_DEG+1);
		cuchebExit(-1);
	}
	
	// check incpts
	if(incpts <= 0){
		fprintf(stderr,"\nIn %s line: %d, incpts must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}

	// check a and b
	if(&a[0] == &b[0]){
		fprintf(stderr,"\nIn %s line: %d, a must != b.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// query device
	int dev;
	hipDeviceProp_t prop;
	cuchebCheckError(hipGetDevice(&dev),__FILE__,__LINE__);
	cuchebCheckError(hipGetDeviceProperties(&prop,dev),__FILE__,__LINE__);
	
	// set blockSize
	int blockSize;
	blockSize = prop.maxThreadsPerBlock;
	
	// set gridSize
	int gridSize;
	gridSize = (int)ceil((double)n/blockSize);
	
	// launch kernel
	zpoints<<<gridSize,blockSize>>>(n,a,b,pts,incpts);
	
	// check for kernel error
	cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
	// return success
	return CUCHEB_STATUS_SUCCESS;
}
