#include <cucheb.h>

/* routine to free memory in cuchebblocklanczos object */
int cuchebblocklanczos_destroy(cuchebblocklanczos* ccb){

  // free index
  delete[] ccb->index;

  // free bands 
  delete[] ccb->bands;

  // free evals
  delete[] ccb->evals;

  // free res
  delete[] ccb->res;

  // free schurvecs
  delete[] ccb->schurvecs;

  // free dtemp
  hipFree(ccb->dtemp);

  // free dvecs
  hipFree(ccb->dvecs);

  // free dschurvecs
  hipFree(ccb->dschurvecs);

  // return  
  return 0;

}
