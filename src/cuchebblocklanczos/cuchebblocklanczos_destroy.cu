#include <cucheb.h>

/* routine to free memory in cuchebblocklanczos object */
int cuchebblocklanczos_destroy(cuchebblocklanczos* ccb){

  // free index
  delete[] ccb->index;

  // free bands 
  delete[] ccb->bands;

  // free evals
  delete[] ccb->evals;

  // free schurvecs
  delete[] ccb->schurvecs;

  // free dtemp
  hipFree(ccb->dtemp);

  // free dvecs
  hipFree(ccb->dvecs);

  // free dschurvecs
  hipFree(ccb->dschurvecs);

  // return  
  return 0;

}
