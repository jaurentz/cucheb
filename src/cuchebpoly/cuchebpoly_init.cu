#include <cucheb.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/* routine for basic initialization */
int cuchebpoly_init(cuchebpoly* ccp){

  // set degree
  ccp->degree = 0;
  
  // set a and b
  ccp->a = -1.0;
  ccp->b = 1.0;
  
  // initialize hipfftHandle
  hipfftPlan1d(&(ccp->cuffthandle), 2*MAX_DOUBLE_DEG, HIPFFT_D2Z, 1);

size_t freeMem, totalMem;
hipMemGetInfo(&freeMem, &totalMem);
printf("cuchebpoly_init\n");
printf("Free = %ld, Total = %ld\n", freeMem, totalMem);

  // allocate workspace
//  if(hipMalloc(&(ccp->dinput),2*MAX_DOUBLE_DEG*sizeof(hipfftDoubleReal)) != 0) {
//    printf("Device memory allocation failed: dinput\n");
gpuErrchk(hipMalloc(&(ccp->dinput),2*MAX_DOUBLE_DEG*sizeof(hipfftDoubleReal)));
//    exit(1);
//  }
  if(hipMalloc(&(ccp->doutput),(MAX_DOUBLE_DEG+1)*sizeof(hipfftDoubleComplex)) != 0) {
    printf("Device memory allocation failed: doutput\n");
    exit(1);
  }

hipMemGetInfo(&freeMem, &totalMem);
printf("Free = %ld, Total = %ld\n", freeMem, totalMem);
 
  // return 
  return 0;

}

