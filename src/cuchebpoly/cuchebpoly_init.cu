#include <cucheb.h>

/* routine for basic initialization */
int cuchebpoly_init(cuchebpoly* ccp){

  // set degree
  ccp->degree = 0;
  
  // set a and b
  ccp->a = -1.0;
  ccp->b = 1.0;
  
  // initialize hipfftHandle
  hipfftPlan1d(&(ccp->cuffthandle), 2*MAX_DOUBLE_DEG, HIPFFT_D2Z, 1);

  // allocate workspace
  if(hipMalloc(&(ccp->dinput),2*MAX_DOUBLE_DEG*sizeof(hipfftDoubleReal)) != 0) {
    printf("Memory allocation failed.\n");
    exit(1);
  }
  if(hipMalloc(&(ccp->doutput),(MAX_DOUBLE_DEG+1)*sizeof(hipfftDoubleComplex)) != 0) {
    printf("Memory allocation failed.\n");
    exit(1);
  }
 
  // return 
  return 0;

}

