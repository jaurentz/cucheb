#include <cuchebpoly.h>

/* routine for destroying cuchebpoly object */
int cuchebpoly_destroy(cuchebpoly* ccp){

  // free cufft
  hipfftDestroy(ccp->cuffthandle);
 
  // free workspace
  hipFree(ccp->dinput);
  hipFree(ccp->doutput);
 
  // return 
  return 0;

}

