#include <cucheb.h>

cuchebStatus_t cuchebDeigs(cuchebLanczosHandle* LH, cuchebOpMult OPMULT, void* USERDATA, double *eigvecs){
	// check n
	int n = LH->n;
	if(n < 1){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check numeigs
	int numeigs = LH->numeigs;
	if(numeigs < 1 || numeigs > n){
		fprintf(stderr,"\nIn %s line: %d, numeigs must be > 0 and <= n.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check runlength
	int runlength = LH->runlength;
	if(runlength < 1 || runlength >= n){
		fprintf(stderr,"\nIn %s line: %d, runlength must be > 0 and < n.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check restarts
	int restarts = LH->restarts;
	if(restarts < 0){
		fprintf(stderr,"\nIn %s line: %d, restarts must be => 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check tol
	double tol = LH->tol;
	if(tol <= 0){
		fprintf(stderr,"\nIn %s line: %d, tol must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	if(tol < DBL_EPSILON){
		fprintf(stderr,"\nIn %s line: %d, tol is below machine precision. Algorithm may not converge.\n",__FILE__,__LINE__);
	}

	// check numconv
	int numconv = LH->numconv;
	if(numconv < 0){numconv = 0;}
	if(numconv >= numeigs){return CUCHEB_STATUS_SUCCESS;}
	
	// allocate memory for Lanzcos
	double *vecs, *diags, *sdiags, *ritzvecs;
	cuchebCheckError(hipMalloc(&vecs,(runlength+1)*n*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&diags,runlength*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&sdiags,runlength*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&ritzvecs,(numeigs+1)*n*sizeof(double)),__FILE__,__LINE__);
	
	// initialize cublas
	hipblasHandle_t cublas_handle;
	cuchebCheckError(hipblasCreate(&cublas_handle),__FILE__,__LINE__);
	cuchebCheckError(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST),__FILE__,__LINE__);
	
	// check starting vector
	double temp;
	hiprandGenerator_t curand_gen;
	cuchebCheckError(hipblasDnrm2(cublas_handle,n,eigvecs,1,&temp),__FILE__,__LINE__);
	if(temp >= 0.0){
		temp = 1.0/temp;
		cuchebCheckError(hipblasDscal(cublas_handle,n,&temp,eigvecs,1),__FILE__,__LINE__);
	}
	else{
		// initialize hiprand
		cuchebCheckError(hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT),__FILE__,__LINE__);
		cuchebCheckError(hiprandSetPseudoRandomGeneratorSeed(curand_gen,time(NULL)),__FILE__,__LINE__);
	
		// random starting vector
		cuchebCheckError(hiprandGenerateNormalDouble(curand_gen,vecs,n,0.0,1.0),__FILE__,__LINE__);
		cuchebCheckError(hipblasDnrm2(cublas_handle,n,vecs,1,&temp),__FILE__,__LINE__);
		temp = 1.0/temp;
		cuchebCheckError(hipblasDscal(cublas_handle,n,&temp,vecs,1),__FILE__,__LINE__);	
		
		// shutdown hiprand
		cuchebCheckError(hiprandDestroyGenerator(curand_gen),__FILE__,__LINE__);
	}
	cuchebCheckError(hipMemcpy(vecs,eigvecs,n*sizeof(double),hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	// lanczos
	int nummatvecs = 0;
	for(int ii=0;ii<restarts+1;ii++){
	
		// do Lanzcos run
		cuchebCheckError(cuchebDlanczos(n,OPMULT,USERDATA,numconv,runlength,vecs,diags,sdiags),__FILE__,__LINE__);
		
		// update nummatvecs
		nummatvecs += runlength-numconv;

		// restart
		cuchebCheckError(cuchebDrestart(n,runlength,numeigs,&numconv,vecs,diags,sdiags,ritzvecs,tol),__FILE__,__LINE__);

		// check convergence
		if(numconv == numeigs){
			LH->numconv = numconv;
			LH->numrestarts = ii;
			LH->nummatvecs = nummatvecs;
			break;
		}
		
		// check iterations
		if(ii == restarts){
			LH->numconv = numconv;
			LH->numrestarts = ii;
			LH->nummatvecs = nummatvecs;
		}
	}	
			
	// shutdown cublas
	cuchebCheckError(hipblasDestroy(cublas_handle),__FILE__,__LINE__);
	
	// copy ritzvecs into eigvecs
	cuchebCheckError(hipMemcpy(eigvecs,ritzvecs,numeigs*n*sizeof(double),hipMemcpyDeviceToHost),__FILE__,__LINE__);

	// free memory
	cuchebCheckError(hipFree(vecs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(diags),__FILE__,__LINE__);
	cuchebCheckError(hipFree(sdiags),__FILE__,__LINE__);
	cuchebCheckError(hipFree(ritzvecs),__FILE__,__LINE__);

	// return success
	return CUCHEB_STATUS_SUCCESS;
}

cuchebStatus_t cuchebDeigs(cuchebLanczosHandle* LH, ChebOp* CO, double *eigvecs){

	// check n
	int n = LH->n;
	if(n < 1){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check numeigs
	int numeigs = LH->numeigs;
	if(numeigs < 1 || numeigs > n){
		fprintf(stderr,"\nIn %s line: %d, numeigs must be > 0 and <= n.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check runlength
	int runlength = LH->runlength;
	if(runlength < 1 || runlength >= n){
		fprintf(stderr,"\nIn %s line: %d, runlength must be > 0 and < n.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check restarts
	int restarts = LH->restarts;
	if(restarts < 0){
		fprintf(stderr,"\nIn %s line: %d, restarts must be => 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check tol
	double tol = LH->tol;
	if(tol <= 0){
		fprintf(stderr,"\nIn %s line: %d, tol must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	if(tol < DBL_EPSILON){
		fprintf(stderr,"\nIn %s line: %d, tol is below machine precision. Algorithm may not converge.\n",__FILE__,__LINE__);
	}

	// check numconv
	int numconv = LH->numconv;
	if(numconv < 0){numconv = 0;}
	if(numconv >= numeigs){return CUCHEB_STATUS_SUCCESS;}

	// allocate memory for Lanzcos
	double *vecs, *diags, *sdiags, *ritzvecs;
	cuchebCheckError(hipMalloc(&vecs,(runlength+1)*n*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&diags,runlength*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&sdiags,runlength*sizeof(double)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&ritzvecs,(numeigs+1)*n*sizeof(double)),__FILE__,__LINE__);
	
	// initialize cublas
	hipblasHandle_t cublas_handle;
	cuchebCheckError(hipblasCreate(&cublas_handle),__FILE__,__LINE__);
	cuchebCheckError(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST),__FILE__,__LINE__);
	
	// check starting vector
	double temp;
	hiprandGenerator_t curand_gen;
	cuchebCheckError(hipblasDnrm2(cublas_handle,n,eigvecs,1,&temp),__FILE__,__LINE__);
	if(temp >= 0.0){
		temp = 1.0/temp;
		cuchebCheckError(hipblasDscal(cublas_handle,n,&temp,eigvecs,1),__FILE__,__LINE__);
	}
	else{
		// initialize hiprand
		cuchebCheckError(hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT),__FILE__,__LINE__);
		cuchebCheckError(hiprandSetPseudoRandomGeneratorSeed(curand_gen,time(NULL)),__FILE__,__LINE__);
	
		// random starting vector
		cuchebCheckError(hiprandGenerateNormalDouble(curand_gen,vecs,n,0.0,1.0),__FILE__,__LINE__);
		cuchebCheckError(hipblasDnrm2(cublas_handle,n,vecs,1,&temp),__FILE__,__LINE__);
		temp = 1.0/temp;
		cuchebCheckError(hipblasDscal(cublas_handle,n,&temp,vecs,1),__FILE__,__LINE__);	
		
		// shutdown hiprand
		cuchebCheckError(hiprandDestroyGenerator(curand_gen),__FILE__,__LINE__);
	}
	cuchebCheckError(hipMemcpy(vecs,eigvecs,n*sizeof(double),hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	
	// lanczos
	int nummatvecs = 0;
	for(int ii=0;ii<restarts+1;ii++){
	
		// do Lanzcos run
		cuchebCheckError(cuchebDlanczos(CO,numconv,runlength,vecs,diags,sdiags),__FILE__,__LINE__);
		
		// update nummatvecs
		nummatvecs += (runlength-numconv)*(CO->getChebpoly()->getDegree());

		// restart
		cuchebCheckError(cuchebDrestart(n,runlength,numeigs,&numconv,vecs,diags,sdiags,ritzvecs,tol),__FILE__,__LINE__);

		// check convergence
		if(numconv == numeigs){
			LH->numconv = numconv;
			LH->numrestarts = ii;
			LH->nummatvecs = nummatvecs;
			break;
		}
		
		// check iterations
		if(ii == restarts){
			LH->numconv = numconv;
			LH->numrestarts = ii;
			LH->nummatvecs = nummatvecs;
		}
	}	
			
	// shutdown cublas
	cuchebCheckError(hipblasDestroy(cublas_handle),__FILE__,__LINE__);
	
	// copy ritzvecs into eigvecs
	cuchebCheckError(hipMemcpy(eigvecs,ritzvecs,numeigs*n*sizeof(double),hipMemcpyDeviceToDevice),__FILE__,__LINE__);

	// free memory
	cuchebCheckError(hipFree(vecs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(diags),__FILE__,__LINE__);
	cuchebCheckError(hipFree(sdiags),__FILE__,__LINE__);
	cuchebCheckError(hipFree(ritzvecs),__FILE__,__LINE__);

	// return success
	return CUCHEB_STATUS_SUCCESS;
}
