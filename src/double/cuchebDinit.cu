#include "hip/hip_runtime.h"
#include <cucheb.h>

__global__ void dinit(int n,double *x,int incx,double val){
	int ii = (blockIdx.z*gridDim.y*gridDim.x + blockIdx.y*gridDim.x + blockIdx.x)*blockDim.x*blockDim.y*blockDim.z 
			+ threadIdx.z*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x;

	if(ii < n){
		x[ii*incx] = val;
	}
}

cuchebStatus_t cuchebDinit(int n,double *x,int incx,double val){

	// check n
	if(n <= 0){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check incx
	if(incx <= 0){
		fprintf(stderr,"\nIn %s line: %d, incx must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// set blockSize and gridsize
	dim3 blockSize, gridSize;
	cuchebCheckError(cuchebSetGridBlocks(n,&blockSize,&gridSize),__FILE__,__LINE__);

	// call kernel
	dinit<<<gridSize,blockSize>>>(n,x,incx,val);
	//dinit<<<1,n>>>(n,x,incx,val);
	
	// check for kernel error
	cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
	// return
	return CUCHEB_STATUS_SUCCESS;
}
