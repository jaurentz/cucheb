#include "hip/hip_runtime.h"
#include <gpusollib.h>

void filtered_spmv_csr_vector(matrix_t *mat, double *x, double *y, int degree, double w, double c, double *d_mu, int mat_choice) {

  // Initialization and declaration  
  csr_t  *csr = mat->d_csr;  // take csr format of matrix A
  int     n   = csr->n;      // take size of matrix A
  int     nnz = csr->nnz;    // take nonzero entries of matrix A
  double  scal, one = 1.0, zero = 0.0, minusc = 0.0;
  double  *vkm1, *vk, *Avk;
  double  minusone = -1.0;
  int     i, k;

  minusc = -c;

  hipsparseMatDescr_t descra;
  hipsparseCreateMatDescr(&descra);
  hipsparseSetMatType(descra, HIPSPARSE_MATRIX_TYPE_GENERAL);

  // For the CUSPARSE, CUBLAS contexts
  hipsparseHandle_t cusparseHandle = 0;
  hipsparseStatus_t cusparseStatus;
  cusparseStatus = hipsparseCreate(&cusparseHandle);

  /*
  if (checkCudaErrors(cusparseStatus))
  {
     exit(EXIT_FAILURE);
  }
  */

  hipblasHandle_t cublasHandle = 0;
  hipblasStatus_t hipblasStatus_t;
  hipblasStatus_t = hipblasCreate(&cublasHandle);

  /*
  if (checkCudaErrors(hipblasStatus_t))
  {
    exit(EXIT_FAILURE);
  }
  */

  /*
  int hwb  = BLOCKDIM / HALFWARP; // Determine number of half-warps per block
  int gDim = min(MAXTHREADS / BLOCKDIM, (n+hwb-1) / hwb);
  int bDim = BLOCKDIM;
  */


  // allocate memory
  hipMalloc( (double**)&vkm1, n*sizeof(double) );
  hipMalloc( (double**)&vk,   n*sizeof(double) );
  hipMalloc( (double**)&Avk,  n*sizeof(double) );

  // initialize buffers
  hipMemset( vkm1, 0.0, n*sizeof(double) );
  hipMemset( vk, 0.0,   n*sizeof(double) );
  hipMemset( Avk, 0.0,  n*sizeof(double) );
  hipMemset( y, 0.0,    n*sizeof(double) );

  // copy x to vk
  hipMemcpy( vk, x, n*sizeof(double), hipMemcpyDeviceToDevice );


double *temp = (double*) malloc((nnz)*sizeof(double));


  for (k = 0; k <= degree; k++) {
   
     hipblasDaxpy(cublasHandle, n, &d_mu[k], vk, 1, y, 1);

     scal = 2.0 / w;
     if (k==0)
        scal = 1.0 / w;

     if ( mat_choice == 0 ) {
        hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descra, csr->a, csr->ia, csr->ja, vk, &zero, Avk);
     }

     hipblasDaxpy(cublasHandle, n, &minusc, vk, 1, Avk, 1);
     hipblasDscal(cublasHandle, n, &scal, Avk, 1);
     hipblasDaxpy(cublasHandle, n, &minusone, vkm1, 1, Avk, 1);

     hipMemcpy( vkm1, vk, n*sizeof(double), hipMemcpyDeviceToDevice );  // vkm1 = vk;
     hipMemcpy( vk, Avk, n*sizeof(double), hipMemcpyDeviceToDevice  );  // vk = vkp1;
     hipMemset( Avk, 0.0, n*sizeof(double) );

  }

  hipsparseDestroy(cusparseHandle);
  hipblasDestroy(cublasHandle);
  hipFree(vk);
  hipFree(vkm1);
  hipFree(Avk);

}










