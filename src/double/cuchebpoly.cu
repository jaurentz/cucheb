#include "hip/hip_runtime.h"
#include <cuchebpoly.h>

/* routine for basic initialization */
int cuchebpoly_init(cuchebpoly* ccp){

  // set degree
  ccp->degree = 0;
  
  // set a and b
  ccp->a = -1.0;
  ccp->b = 1.0;
  
  // set coeffs
  ccp->coeffs[0] = 1.0;
  for (int ii=0; ii<DOUBLE_DEG; ii++) {
    ccp->coeffs[ii+1] = 0.0;
  }

  // return 
  return 0;

}

/* routine for standard print */
int cuchebpoly_print(cuchebpoly* ccp){

  // print banner
  printf("\ncuchebpoly:\n");

  // degree
  printf(" degree = %d\n",ccp->degree);
 
  // a and b
  printf(" [a,b] = [%+e,%+e]\n",ccp->a,ccp->b);
  
  // coeffs
  for (int ii=0; ii<ccp->degree+1; ii++) {
    printf(" coeffs[%d] = %+e\n",ii,ccp->coeffs[ii]);
  }
  printf("\n");

  // return 
  return 0;

}

/* routine for long print */
int cuchebpoly_printlong(cuchebpoly* ccp){

  // print banner
  printf("\ncuchebpoly:\n");

  // degree
  printf(" degree = %d\n",ccp->degree);
 
  // a and b
  printf(" [a,b] = [%+e,%+e]\n",ccp->a,ccp->b);
  
  // coeffs
  for (int ii=0; ii<DOUBLE_DEG+1; ii++) {
    printf(" coeffs[%d] = %+e\n",ii,ccp->coeffs[ii]);
  }
  printf("\n");

  // return 
  return 0;

}

/* routine for Chebyshev points */
int cuchebpoints(double a, double b, double* points){

  // check a and b
  if ( a >= b ) {
    return 1;
  }

  // set points
  double alpha = (b-a)/2.0;
  double beta = (b+a)/2.0;
  for (int ii=0; ii<DOUBLE_DEG+1; ii++) {
    points[ii] = alpha*sin(DOUBLE_PI*(2.0*ii-DOUBLE_DEG)/(2.0*DOUBLE_DEG)) +
                  beta;
  }

  // return 
  return 0;

}

/* cuchebcoeffs */
int cuchebcoeffs (double *coeffs){
 
  // allocate workspace
  hipfftDoubleReal *input;
  hipMalloc(&input,2*DOUBLE_DEG*sizeof(hipfftDoubleReal));
  hipfftDoubleComplex *output;
  hipMalloc(&output,(DOUBLE_DEG+1)*sizeof(hipfftDoubleComplex));
 
  // initialize cufft
  hipfftHandle cufftHand;
  hipfftPlan1d(&cufftHand, 2*DOUBLE_DEG, HIPFFT_D2Z, 1);
 
  // initialize input 
  int deg = DOUBLE_DEG;
  int N = 2*deg;

  hipMemcpy(&input[0], &coeffs[deg], sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(&input[deg], &coeffs[0], sizeof(double), hipMemcpyHostToDevice);
  for (int ii=1; ii<deg; ii++) {
    hipMemcpy(&input[ii], &coeffs[deg-ii], sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&input[N-ii], &coeffs[deg-ii], sizeof(double), hipMemcpyHostToDevice);
  }

  // execute plan
  hipfftExecD2Z(cufftHand,input,output);
 
  // extract output
  hipMemcpy(&coeffs[0], &output[0], sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&coeffs[deg], &output[deg], sizeof(double), hipMemcpyDeviceToHost);
  for (int ii=1; ii<deg; ii++) {
    hipMemcpy(&coeffs[ii], &output[ii], sizeof(double), hipMemcpyDeviceToHost);
  }

  // normalize output
  coeffs[0] = coeffs[0]/(double)(deg)/2.0;
  coeffs[deg] = coeffs[deg]/(double)(deg)/2.0;
  for (int ii=1; ii<deg; ii++) {
    coeffs[ii] = coeffs[ii]/(double)(deg);
  }

  // free cufft
  hipfftDestroy(cufftHand);
 
  // free workspace
  hipFree(input);
  hipFree(output);
 
  // return success
  return 0;

}

/* routine for chopping Chebyshev coefficients */
int cuchebchop(int* degree, double* coeffs){

  // find maximum
  double maximum = 0.0;
  for (int ii=0; ii<DOUBLE_DEG+1; ii++) {
    maximum = max(maximum, abs(coeffs[ii]));
  }

  // maximum == 0
  if (maximum == 0) {
    *degree = 0;
  }
  
  // maximum != 0
  else {
    
    // compute degree
    for (int ii=0; ii<MAX_DOUBLE_DEG+1; ii++) {

      // set current degree
      *degree = MAX_DOUBLE_DEG-ii;

      // exit if trailing coefficient is too large
      if (abs(coeffs[MAX_DOUBLE_DEG-ii]) >= DOUBLE_EPS*maximum) {
        break;
      }

    }

  }

  // return 
  return 0;

}

/* routine for creating point filter */
int cuchebpoly_pointfilter(double a, double b, double rho, cuchebpoly* ccp){

  // check a and b
  if ( a >= b ) {
    return 1;
  }

  // set a and b in ccp
  ccp->a = a;
  ccp->b = b;

  // compute Chebyshev points in [a,b]
  cuchebpoints(a,b,&(ccp->coeffs[0]));

  // compute shift
  double shift;
  if (rho <= a) {shift = a;}
  else if (rho >= b) {shift = b;}
  else {shift = rho;}

  // compute function values for f(x) = exp(-100*(x-shift)^2)
  double scl = pow(b - a,2);
  for (int ii=0; ii<DOUBLE_DEG+1; ii++) {
    ccp->coeffs[ii] = exp(-100.0*pow(ccp->coeffs[ii]-shift,2)/scl);
  }
 
  // compute Chebyshev coefficients
  cuchebcoeffs(&(ccp->coeffs[0]));

  // chop Chebyshev coefficients
  cuchebchop(&(ccp->degree),&(ccp->coeffs[0]));

  // return 
  return 0;

}

/* routine for creating step filter */
int cuchebpoly_stepfilter(double a, double b, double c, double d, cuchebpoly* ccp){

  // check a and b
  if ( a >= b ) {
    return 1;
  }

  // check c and d
  if ( c >= d ) {
    return 1;
  }

  // compute lower bound 
  double lb;
  if (c <= a) {lb = a;}
  else if (c >= b) {return 1;}
  else {lb = c;}

  // compute upper bound 
  double ub;
  if (d >= b) {ub = b;}
  else if (d <= a) {return 1;}
  else {ub = d;}

  // set degree
  ccp->degree = MAX_DOUBLE_DEG;

  // set a and b in ccp
  ccp->a = a;
  ccp->b = b;

  // scale everything to [-1,1]
  lb = (2.0*lb - (b+a))/(b-a);
  ub = (2.0*ub - (b+a))/(b-a);
 
  // compute Chebyshev coefficients
  double pi = DOUBLE_PI;
  double aclb = acos(lb);
  double acub = acos(ub);
  ccp->coeffs[0] = (aclb - acub)/pi;
  for (int ii=1; ii<MAX_DOUBLE_DEG+1; ii++) {
    ccp->coeffs[ii] = 2.0*(sin(ii*aclb) - sin(ii*acub))/(ii*pi);
  }

  // apply Jackson damping
  int deg = MAX_DOUBLE_DEG;
  double alpha = 1.0/(deg+2.0);
  double beta = sin(pi*alpha);
  double gamma = cos(pi*alpha);
  for (int ii=0; ii<MAX_DOUBLE_DEG+1; ii++) {
    ccp->coeffs[ii] = alpha*((deg+2.0-ii)*beta*cos(ii*pi*alpha) +
                       sin(ii*pi*alpha)*gamma)*ccp->coeffs[ii]/beta;
  }

  // chop Chebyshev coefficients
  cuchebchop(&(ccp->degree),&(ccp->coeffs[0]));

  // return 
  return 0;

}

