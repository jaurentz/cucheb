#include "hip/hip_runtime.h"
#include <cuchebpoly.h>

/* routine for basic initialization */
int cuchebpoly_init(cuchebpoly* ccp){

  // set degree
  ccp->degree = 0;
  
  // set a and b
  ccp->a = -1.0;
  ccp->b = 1.0;
  
  // set coeffs
  ccp->coeffs[0] = 1.0;
  for (int ii=0; ii<DOUBLE_DEG; ii++) {
    ccp->coeffs[ii+1] = 0.0;
  }

  // return 
  return 0;

}

/* routine for standard print */
int cuchebpoly_print(cuchebpoly* ccp){

  // print banner
  printf("\ncuchebpoly:\n");

  // degree
  printf(" degree = %d\n",ccp->degree);
 
  // a and b
  printf(" [a,b] = [%+e,%+e]\n",ccp->a,ccp->b);
  
  // coeffs
  for (int ii=0; ii<ccp->degree+1; ii++) {
    printf(" coeffs[%d] = %+e\n",ii,ccp->coeffs[ii]);
  }
  printf("\n");

  // return 
  return 0;

}

/* routine for long print */
int cuchebpoly_printlong(cuchebpoly* ccp){

  // print banner
  printf("\ncuchebpoly:\n");

  // degree
  printf(" degree = %d\n",ccp->degree);
 
  // a and b
  printf(" [a,b] = [%+e,%+e]\n",ccp->a,ccp->b);
  
  // coeffs
  for (int ii=0; ii<DOUBLE_DEG+1; ii++) {
    printf(" coeffs[%d] = %+e\n",ii,ccp->coeffs[ii]);
  }
  printf("\n");

  // return 
  return 0;

}

/* routine for Chebyshev points */
int cuchebpoints(double a, double b, double* points){

  // check a and b
  if ( a >= b ) {
    return 1;
  }

  // set points
  double alpha = (b-a)/2.0;
  double beta = (b+a)/2.0;
  for (int ii=0; ii<DOUBLE_DEG+1; ii++) {
    points[ii] = alpha*sin(DOUBLE_PI*(2.0*ii-DOUBLE_DEG)/(2.0*DOUBLE_DEG)) +
                  beta;
  }

  // return 
  return 0;

}

/* cuchebcoeffs */
int cuchebcoeffs (double *coeffs){
 
  // allocate workspace
  hipfftDoubleReal *input;
  hipMalloc(&input,2*DOUBLE_DEG*sizeof(hipfftDoubleReal));
  hipfftDoubleComplex *output;
  hipMalloc(&output,(DOUBLE_DEG+1)*sizeof(hipfftDoubleComplex));
 
  // initialize cufft
  hipfftHandle cufftHand;
  hipfftPlan1d(&cufftHand, 2*DOUBLE_DEG, HIPFFT_D2Z, 1);
 
  // initialize input 
  int deg = DOUBLE_DEG;
  int N = 2*deg;

  hipMemcpy(&coeffs[deg], &input[0], sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(&coeffs[0], &input[deg], sizeof(double), hipMemcpyHostToDevice);
  for (int ii=1; ii<deg; ii++) {
    hipMemcpy(&coeffs[deg-ii], &input[ii], sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&coeffs[deg-ii], &input[N-ii], sizeof(double), hipMemcpyHostToDevice);
  }
 
  // execute plan
  hipfftExecD2Z(cufftHand,input,output);
 
  // extract output
  hipMemcpy(&output[deg], &coeffs[0], sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&output[0], &coeffs[deg], sizeof(double), hipMemcpyDeviceToHost);
  for (int ii=1; ii<deg; ii++) {
    hipMemcpy(&output[deg-ii], &coeffs[ii], sizeof(double), hipMemcpyDeviceToHost);
  }

  // normalize output
  coeffs[0] = coeffs[0]/(double)(deg)/2.0;
  coeffs[deg] = coeffs[deg]/(double)(deg)/2.0;
  for (int ii=1; ii<deg; ii++) {
    coeffs[ii] = coeffs[ii]/(double)(deg);
  }

  // free cufft
  hipfftDestroy(cufftHand);
 
  // free workspace
  hipFree(input);
  hipFree(output);
 
  // return success
  return 0;

}
