#include <cuchebmatrix.h>

/* routine to free memory in cuchebmatrix object */
int cuchebmatrix_destroy(cuchebmatrix* ccm){

  // free rowinds
  delete[] ccm->rowinds;

  // free colinds
  delete[] ccm->colinds;

  // free vals
  delete[] ccm->vals;
 
  // return  
  return 0;

}
