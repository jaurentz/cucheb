#include "hip/hip_runtime.h"

#include "gpusollib.h"
#define  SEED 100


void filtered_lanczos(matrix_t *mat, int msteps, int degree, double w, double c, int mat_choice) {


   //-------------------
   // Declare variables
   //------------------
   int     i, n;
   double  *h_v, *d_w, *d_u, *VV, *d, *e, *z, *work, *temp;
   double  t, dummy, beta, minusbeta, alpha, minusalpha, orthTol, wn, one = 1.0, zero = 0.0, minusone = -1.0;
   double  alpha2, beta2, t2, dummy2, minusalpha2, minusbeta2, d_one, d_zero, d_minusone;

   hipblasHandle_t cublasHandle = 0;
   hipblasStatus_t hipblasStatus_t;
   hipblasStatus_t = hipblasCreate(&cublasHandle);

   //-----------------------------------
   // Initialize random number generator
   //-----------------------------------
   srand(SEED);

   // Number of rows
   n = mat->n;

   // Generate initial random vector for Lanczos
   h_v = (double*) malloc( n * sizeof(double) );
   for (i = 0; i < n; i++) {
      h_v[i] = rand()/((double)RAND_MAX + 1);
   }

   //---------------------------------------------------------
   // Set up device memory for Lanczos basis and other vectors
   //---------------------------------------------------------
   hipMalloc( (void**)&VV, (msteps+1) * n * sizeof(double) );
   hipMemset(VV, 0.0, (msteps+1) * n * sizeof(double) );

   hipMalloc( (void**)&d_w, n * sizeof(double) );
   hipMemset(d_w, 0.0, n * sizeof(double) );

   hipMalloc( (void**)&d_u, msteps * sizeof(double) );
   hipMemset(d_u, 0.0, n * sizeof(double) );

   hipMalloc( (void**)&temp, n * sizeof(double) );
   hipMemset(temp, 0.0, n * sizeof(double) );

   // Scalar values that reside in the GPU
   hipMalloc( (void**)&alpha2, sizeof(double) );
   hipMemset(&alpha2, 0.0, sizeof(double) );
   hipMalloc( (void**)&beta2, sizeof(double) );
   hipMemset(&beta2, 0.0, sizeof(double) );
   hipMalloc( (void**)&t2, sizeof(double) );
   hipMemset(&t2, 0.0, sizeof(double) );
   hipMalloc( (void**)&minusalpha2, sizeof(double) );
   hipMemset(&minusalpha2, 0.0, sizeof(double) );
   hipMalloc( (void**)&minusbeta2, sizeof(double) );
   hipMemset(&minusbeta2, 0.0, sizeof(double) );
   hipMalloc( (void**)&dummy2, sizeof(double) );
   hipMemset(&dummy2, 0.0, sizeof(double) );
   hipMalloc( (void**)&d_one, sizeof(double) );
   hipMemset(&d_one, 1.0, sizeof(double) );
   hipMalloc( (void**)&d_zero, sizeof(double) );
   hipMemset(&d_zero, 0.0, sizeof(double) );
   hipMalloc( (void**)&d_minusone, sizeof(double) );
   hipMemset(&d_minusone, -1.0, sizeof(double) );

   // Scale starting Lanczos vector
   hipMemcpy( VV, h_v, n * sizeof(double), hipMemcpyHostToDevice );
   hipblasDnrm2(cublasHandle, n, VV, 1, &t2);
   hipMemcpy( &t, &t2, n * sizeof(double), hipMemcpyDeviceToHost );
   dummy = 1.0 / t;
   hipMemcpy( &dummy2, &dummy, n * sizeof(double), hipMemcpyHostToDevice );
   hipblasDscal(cublasHandle, n, &dummy2, VV, 1);

   // set up other initial variables
   beta    = 0.0;
   alpha   = 0.0;
   orthTol = 1.0e-8; // just some checking
   wn      = 0.0;    //    >>


   //------------------------------------------------------
   // Allocate space to hold the tridiagonal Lanczos matrix
   // -----------------------------------------------------
   d = (double*) malloc( msteps * sizeof(double) );
   e = (double*) malloc( (msteps-1) * sizeof(double) );


   //---------------------------------
   // Description of filter parameters
   //---------------------------------
   // damping  : Jackson, Jackson-Chebychev, Delta 
   // xi       : test vector to check the quality of the polynomial
   // [i1, i2] : After scaling the eigvals of A, we seek the 
   //            eigenvalues in [i1,i2]
   // mu       : the coefficients
   // d_mu     : the coefficients' copy in the device

   int     damping;
   double  i1, i2;
   double  *mu, *d_mu;
   i1 = 0.15;
   i2 = 0.20;
   damping = 0;
   mu = (double*) malloc( (degree + 1) * sizeof(double) );
   memset (mu, 0.0, (degree + 1) * sizeof(double) );


   //--------------------------------------------------------
   // Compute polynomial coefficients and copy them to device
   compute_coeff(degree, i1, i2, damping, mu);
   hipMalloc( (void**)&d_mu, (degree+1) * sizeof(double) );
   hipMemcpy( d_mu, mu, (degree+1)*sizeof(double), hipMemcpyHostToDevice );
  
   //--------------------------
   // Lanczos phase has started
   //--------------------------
   printf("Lanczos Alg begins ...\n");

   for ( i = 0; i < msteps; i++) {
     //-------------------
     // MV -- d_w = A*temp
     //-------------------
     hipMemcpy(temp, &VV[i*n], n * sizeof(double), hipMemcpyDeviceToDevice);
     filtered_spmv_csr_vector(mat, temp, d_w, 0, degree, w, c, d_mu, mat_choice); 

     //------------------
     // 3-term recurrence
     //------------------
     minusbeta = -beta;
     hipMemcpy( &minusbeta2, &minusbeta, sizeof(double), hipMemcpyHostToDevice );
     if (i == 0)
        hipblasDaxpy(cublasHandle, n, &minusbeta2, VV, 1, d_w, 1);
     else
        hipblasDaxpy(cublasHandle, n, &minusbeta2, &VV[(i-1)*n], 1, d_w, 1);

     // Compute alpha
     hipblasDdot (cublasHandle, n, d_w, 1, &VV[i*n], 1, &alpha2);
     hipMemcpy( &alpha, &alpha2, sizeof(double), hipMemcpyDeviceToHost );
     minusalpha = -alpha;
     hipMemcpy( &minusalpha2, &minusalpha, sizeof(double), hipMemcpyHostToDevice );
     hipblasDaxpy(cublasHandle, n, &minusalpha2, &VV[i*n], 1, d_w, 1);

     // Add on-diagonal entry
     d[i] = alpha;
     wn += alpha*alpha;
 
     //---------------------------
     // Re-orthogonalization phase
     //---------------------------
     // u = V'*w
     hipblasDgemv(cublasHandle, HIPBLAS_OP_T, n, i+1, &d_one, VV, n, d_w, 1, &d_zero, d_u, 1);

     // w = w - V*u
     hipblasDgemv(cublasHandle, HIPBLAS_OP_N, n, i+1, &d_minusone, VV, n, d_u, 1, &d_one, d_w, 1);

     //-----------------------------------------------
     // Take norm of vector after re-orthogonalization
     //-----------------------------------------------
     hipblasDdot (cublasHandle, n, d_w, 1, d_w, 1, &beta2);
     hipMemcpy( &beta, &beta2, sizeof(double), hipMemcpyDeviceToHost );

     if (beta*(i+1) < orthTol*wn)
       break;
     wn += 2.0 * beta;

     //-----------------
     // Normalize vector
     //-----------------
     beta = sqrt(beta);
     dummy = 1.0 / beta;
     hipMemcpy( &dummy2, &dummy, sizeof(double), hipMemcpyHostToDevice );
     hipblasDaxpy(cublasHandle, n, &dummy2, d_w, 1, &VV[(i+1)*n], 1);

     // Add new off-diagonal entry to T
     if (i < msteps-1) {
       e[i] = beta;
     }

   }


   //----------------------
   // Lanczos phase is over
   //----------------------
   printf("Generating the Lanczos basis : Done\n");


   //-----------------------------------------
   // Solve the tridiagonal eigenvalue problem
   //-----------------------------------------
   printf("Lapack STEQR begins ...\n");
   work = (double*) malloc( 2 * (msteps-1) * sizeof(double) );
   Calloc(z, msteps*msteps, double);
   int info;
   char compz = 'I';
   STEQR(&compz, &msteps, d, e, z, &msteps, work, &info); // use BLAS routines
   if (info != 0) {
     printf("LAPACK: FAILED TO FIND EIGENVALUES !!!\n");
     exit(-1);
   }


   //-------------------
   // Deallocate vectors
   //-------------------
   hipFree(VV);
   hipFree(d_mu);
   hipFree(d_w);
   hipFree(d_u);
   hipFree(temp);
   free(h_v);
   free(d);
   free(e);
   free(work);
   free(z);
   free(mu);


}






