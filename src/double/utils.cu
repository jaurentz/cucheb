#include "gpusollib.h"

/*-----------------------------------------------*/
int cuda_init() {
  int deviceCount, dev;
  hipDeviceProp_t deviceProp;
/*-----------------------------------------------*/
  hipGetDeviceCount(&deviceCount);
  printf("=========================================\n");
  if (deviceCount == 0) {
    printf("There is no device supporting CUDA\n");
    return 1;
  }
  dev = 0;
  CUDA_SAFE_CALL(hipSetDevice(dev));
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
  if (deviceProp.major == 9999 && 
      deviceProp.minor == 9999) {
    printf("There is no device supporting CUDA.\n");
    return 1;
  }
  printf("Running on Device %d: \"%s\"\n", dev, deviceProp.name);
  printf("  Major revision number:          %d\n",
         deviceProp.major);
  printf("  Minor revision number:          %d\n",
           deviceProp.minor);
  printf("  Total amount of global memory:  %.2f GB\n",
         deviceProp.totalGlobalMem/1e9);
  printf("=========================================\n");
  return 0;
}

/*----------------------------------------*/
int gpusol_init() {
  if (cuda_init()) {
    printf("lol1");
    return 1;
  }

  if (cublasInit() != HIPBLAS_STATUS_SUCCESS) {
    printf("lol2");
    return 2;
  }
  return 0;
}

/*--------------------------------------------------*/
void cuda_check_err() {
  hipError_t cudaerr = hipGetLastError() ;
  if (cudaerr != hipSuccess) 
    printf("error: %s\n",hipGetErrorString(cudaerr));
}

/*----------------------------------------------*/
int gpusol_finalize() {
/*------ Shut down CUBLAS */
  if (cublasShutdown() != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS shut down FAILED !!\n");
    return 1;
  }
  cuda_check_err();
  return 0;
}

