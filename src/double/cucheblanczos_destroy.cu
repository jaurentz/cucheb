#include <cucheblanczos.h>

/* routine to free memory in cucheblanczos object */
int cucheblanczos_destroy(cucheblanczos* ccl){

  // free diag
  delete[] ccl->diag;

  // free sdiag
  delete[] ccl->sdiag;

  // free schurvecs
  delete[] ccl->schurvecs;

  // destroy cublas handle
  hipblasDestroy(ccl->handle);
 
  // free dvecs
  hipFree(ccl->dvecs);

  // free dschurvecs
  hipFree(ccl->dschurvecs);

  // return  
  return 0;

}
