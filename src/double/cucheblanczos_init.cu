#include "hip/hip_runtime.h"
#include <cucheblanczos.h>

/* routine to initialize cucheblanczos object */
int cucheblanczos_init(cuchebmatrix* ccm, cucheblanczos* ccl){

  // set dimensions
  ccl->n = ccm->m;
  ccl->nvecs = min(ccl->n,100);

  // allocate host memory
  ccl->diag = new double[ccl->nvecs];
  if (ccl->diag == NULL) {
    printf("Memory allocation failed.\n");
    exit(1);
  }
  ccl->sdiag = new double[ccl->nvecs];
  if (ccl->sdiag == NULL) {
    printf("Memory allocation failed.\n");
    exit(1);
  }
  ccl->schurvecs = new double[(ccl->nvecs)*(ccl->nvecs)];
  if (ccl->schurvecs == NULL) {
    printf("Memory allocation failed.\n");
    exit(1);
  }

  // create cublas handle
  if(hipblasCreate(&(ccl->handle)) != 0) {
    printf("CUBLAS initialization failed.\n");
    exit(1);
  }

  // set pointer mode to Host
  hipblasSetPointerMode(ccl->handle,HIPBLAS_POINTER_MODE_HOST);

  // allocate device memory
  if(hipMalloc(&(ccl->dtemp),(ccl->nvecs)*sizeof(double)) != 0) {
    printf("Memory allocation failed.\n");
    exit(1);
  }
  if(hipMalloc(&(ccl->dvecs),(ccl->n)*((ccl->nvecs)+1)*sizeof(double)) != 0) {
    printf("Memory allocation failed.\n");
    exit(1);
  }
  if(hipMalloc(&(ccl->dschurvecs),(ccl->nvecs)*(ccl->nvecs)*sizeof(double)) != 0) {
    printf("Memory allocation failed.\n");
    exit(1);
  }

  // return  
  return 0;

}

