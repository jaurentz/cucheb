#include <cuchebpoly.h>

/* cuchebpoly_coeffs */
int cuchebpoly_coeffs (double *coeffs){
 
  // allocate workspace
  hipfftDoubleReal *input;
  hipMalloc(&input,2*DOUBLE_DEG*sizeof(hipfftDoubleReal));
  hipfftDoubleComplex *output;
  hipMalloc(&output,(DOUBLE_DEG+1)*sizeof(hipfftDoubleComplex));
 
  // initialize cufft
  hipfftHandle cufftHand;
  hipfftPlan1d(&cufftHand, 2*DOUBLE_DEG, HIPFFT_D2Z, 1);
 
  // initialize input 
  int deg = DOUBLE_DEG;
  int N = 2*deg;

  hipMemcpy(&input[0], &coeffs[deg], sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(&input[deg], &coeffs[0], sizeof(double), hipMemcpyHostToDevice);
  for (int ii=1; ii<deg; ii++) {
    hipMemcpy(&input[ii], &coeffs[deg-ii], sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&input[N-ii], &coeffs[deg-ii], sizeof(double), hipMemcpyHostToDevice);
  }

  // execute plan
  hipfftExecD2Z(cufftHand,input,output);
 
  // extract output
  hipMemcpy(&coeffs[0], &output[0], sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&coeffs[deg], &output[deg], sizeof(double), hipMemcpyDeviceToHost);
  for (int ii=1; ii<deg; ii++) {
    hipMemcpy(&coeffs[ii], &output[ii], sizeof(double), hipMemcpyDeviceToHost);
  }

  // normalize output
  coeffs[0] = coeffs[0]/(double)(deg)/2.0;
  coeffs[deg] = coeffs[deg]/(double)(deg)/2.0;
  for (int ii=1; ii<deg; ii++) {
    coeffs[ii] = coeffs[ii]/(double)(deg);
  }

  // free cufft
  hipfftDestroy(cufftHand);
 
  // free workspace
  hipFree(input);
  hipFree(output);
 
  // return success
  return 0;

}

