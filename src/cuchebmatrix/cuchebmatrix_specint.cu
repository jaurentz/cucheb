#include "hip/hip_runtime.h"
#include <cucheb.h>

/* routine to estimate spectral interval */
int cuchebmatrix_specint(cuchebmatrix* ccm){

  // number of arnoldi steps
  int nblocks;
  nblocks = min(ccm->m,MAX_NUM_BLOCKS);

  // create lanczos object
  cucheblanczos ccl;
  cucheblanczos_init(1,nblocks,ccm,&ccl);

  // set starting vector
  cucheblanczos_startvecs(&ccl);

  // arnoldi run
  cucheblanczos_arnoldi(ccm,&ccl);

  // compute ritz values
  cucheblanczos_eig(ccm,&ccl);

  // compute residuals
  ccm->a = (ccl.evals)[nblocks-1];
  ccm->b = (ccl.evals)[0];

  // destroy ccl
  cucheblanczos_destroy(&ccl);

  // return  
  return 0;

}
