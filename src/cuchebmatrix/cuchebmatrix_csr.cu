#include <cuchebmatrix.h>

/* routine for converting to csr format */
int cuchebmatrix_csr(cuchebmatrix* ccm){

  // loop through row inds
  int cind = 0;
  for(int ii=0; ii<(ccm->nnz); ii++){
    if((ccm->rowinds)[ii] > cind){
      cind += 1;
      (ccm->rowinds)[cind] = ii;
    }
  }
  (ccm->rowinds)[ccm->m] = ccm->nnz;

  // copy to device memory
  // rowinds
  hipMemcpy(ccm->drowinds,ccm->rowinds,((ccm->m)+1)*sizeof(int),hipMemcpyHostToDevice);

  // colinds
  hipMemcpy(ccm->dcolinds,ccm->colinds,(ccm->nnz)*sizeof(int),hipMemcpyHostToDevice);

  // vals
  hipMemcpy(ccm->dvals,ccm->vals,(ccm->nnz)*sizeof(double),hipMemcpyHostToDevice);

  // return 
  return 0;

}


