#include <cucheb.h>

/* routine to free memory in cuchebmatrix object */
int cuchebmatrix_destroy(cuchebmatrix* ccm){

  // free rowinds
  delete[] ccm->rowinds;

  // free colinds
  delete[] ccm->colinds;

  // free vals
  delete[] ccm->vals;

  // destroy cublas handle
  hipblasDestroy(ccm->cublashandle);
 
  // destroy cusparse handle
  hipsparseDestroy(ccm->cusparsehandle);
 
  // destroy cusparse matdescr
  hipsparseDestroyMatDescr(ccm->matdescr);
 
  // free drowinds
  hipFree(ccm->drowinds);

  // free dcolinds
  hipFree(ccm->dcolinds);

  // free dvals
  hipFree(ccm->dvals);

  // free dtemp
  hipFree(ccm->dtemp);

  // return  
  return 0;

}
