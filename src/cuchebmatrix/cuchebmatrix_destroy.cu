#include <cuchebmatrix.h>

/* routine to free memory in cuchebmatrix object */
int cuchebmatrix_destroy(cuchebmatrix* ccm){

  // free rowinds
  delete[] ccm->rowinds;

  // free colinds
  delete[] ccm->colinds;

  // free vals
  delete[] ccm->vals;

  // destroy cusparse handle
  hipsparseDestroy(ccm->handle);
 
  // destroy cusparse matdescr
  hipsparseDestroyMatDescr(ccm->matdescr);
 
  // free drowinds
  hipFree(ccm->drowinds);

  // free dcolinds
  hipFree(ccm->dcolinds);

  // free dvals
  hipFree(ccm->dvals);

  // return  
  return 0;

}
