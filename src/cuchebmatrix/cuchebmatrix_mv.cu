#include <cucheb.h>

/* routine for mv multiply on GPU */
int cuchebmatrix_mv(cuchebmatrix* ccm, double* alpha, double* x, double* beta,
                    double* y){

  // hipsparseDcsrmv
  hipsparseDcsrmv(ccm->cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, ccm->m, ccm->n, 
                 ccm->nnz, alpha, ccm->matdescr, ccm->dvals, ccm->drowinds,
                 ccm->dcolinds, x, beta, y);
 
  // return 
  return 0;

}

