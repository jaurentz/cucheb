#include "hip/hip_runtime.h"
#include <cucheb.h>

/* routine for poly mm multiply on GPU */
/* Y = p(A)*X */
int cuchebmatrix_polymm(cuchebmatrix* ccm, cuchebpoly* ccp, int bsize,
                        double* X, double* Y, double* V1, double* V2){

  // local variables
  int n, deg;
  double a, b;
  double* coeffs;
  n = ccm->m;
  deg = ccp->degree;
  a = ccp->a;
  b = ccp->b;
  coeffs = &(ccp->coeffs)[0];

  // scalars
  double zero = 0.0, mone = -1.0;
  double A, B;
  A = 4.0/(b-a);
  B = -2.0*(b+a)/(b-a);

  // initialize Y
  hipblasDcopy(ccm->cublashandle, bsize*n, X, 1, Y, 1);
  hipblasDscal(ccm->cublashandle, bsize*n, &coeffs[deg], Y, 1);
 
  // initialize V1
  hipblasDcopy(ccm->cublashandle, bsize*n, X, 1, V1, 1);
  hipblasDscal(ccm->cublashandle, bsize*n, &zero, V1, 1);
 
clock_t tick;
double cp_time = 0.0;
double mm_time = 0.0;
double ad_time = 0.0;

  // loop for clenshaw
  for(int ii=0; ii<deg; ii++){

tick = clock();
    // copy V1 to V2
    hipblasDcopy(ccm->cublashandle, bsize*n, V1, 1, V2, 1);

    // copy Y to V1
    hipblasDcopy(ccm->cublashandle, bsize*n, Y, 1, V1, 1);
cp_time += (clock()-tick)/((double)CLOCKS_PER_SEC);

tick = clock();
    // scale A and B if ii == deg-1
    if(ii == deg-1){
      A = A/2.0;
      B = B/2.0;
    }

    // apply matrix
    cuchebmatrix_mm(ccm, bsize, &A, V1, &B, Y);
mm_time += (clock()-tick)/((double)CLOCKS_PER_SEC);

tick = clock();
    // add x
    hipblasDaxpy(ccm->cublashandle, bsize*n, &coeffs[deg-ii-1], X, 1, Y, 1);

    // subtract V2
    hipblasDaxpy(ccm->cublashandle, bsize*n, &mone, V2, 1, Y, 1);
ad_time += (clock()-tick)/((double)CLOCKS_PER_SEC);

  }

printf("\ncp_time = %f\n",cp_time);
printf("mm_time = %f\n",mm_time);
printf("ad_time = %f\n\n",ad_time);


  // return 
  return 0;

}

