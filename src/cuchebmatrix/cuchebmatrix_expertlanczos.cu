#include "hip/hip_runtime.h"
#include <cucheb.h>


/* expert lanczos routine for point with statistics */
int cuchebmatrix_expertlanczos(int neig, double shift, int degree,
                                 int bsize, int numvecs, int stepsize, 
                                 cuchebmatrix* ccm, cucheblanczos* ccl, 
                                 cuchebstats* ccstats){

  // initialize ccstats
  ccstats->mat_dim = 0;
  ccstats->mat_nnz = 0;
  ccstats->block_size = 0;
  ccstats->num_blocks = 0;
  ccstats->num_iters = 0;
  ccstats->num_innerprods = 0;
  ccstats->max_degree = 0;
  ccstats->num_matvecs = 0;
  ccstats->specint_time = 0.0;
  ccstats->arnoldi_time = 0.0;
  ccstats->num_conv = 0;
  ccstats->max_res = 0.0;

  // collect some matrix statistics
  ccstats->mat_dim = ccm->m;
  ccstats->mat_nnz = ccm->nnz;

  // timing variables
  time_t start, stop;

  // compute spectral interval
  start = time(0);
  cuchebmatrix_specint(ccm);

  // record compute time
  stop = time(0);
  ccstats->specint_time = difftime(stop,start);
    
  // initialize lanczos object
  cucheblanczos_init(bsize,numvecs,ccm,ccl);

  // collect some lanczos statistics
  ccstats->block_size = ccl->bsize;

  // set starting vector
  cucheblanczos_startvecs(ccl);

  // compute interval
  double a, b, rho;
  a = ccm->a;
  b = ccm->b;
  rho = min(max(a,shift),b);
  double scl;
  scl = abs(b-a)*(10.0*neig*(ccl->bsize))/(ccm->m);
  double lb, ub;
  lb = max(a,rho-scl);
  ub = min(b,rho+scl);

  // initialize filter polynomial
  cuchebpoly ccp;
  cuchebpoly_init(&ccp);

  // create filter polynomial
  if (degree > -1) {
    cuchebpoly_stepfilter(ccm->a,ccm->b,lb,ub,degree,&ccp);
  }
  else {
    cuchebpoly_smartfilter(ccm->a,ccm->b,lb,ub,&ccp);
  }

  // max_degree
  ccstats->max_degree = max(ccstats->max_degree,ccp.degree);

  // start stop watch
  start = time(0);

  // loop through various Krylov subspaces
  int nres;
  int step;
  step = min(max(stepsize,1),ccl->nblocks);
  nres = (ccl->nblocks)/(step) + 1;
  for (int jj=0; jj<nres; jj++) {

    // filtered arnoldi run
    cucheblanczos_filteredarnoldi(step,ccm,&ccp,ccl,ccstats);

    // update ccstats
    // num_iters
    ccstats->num_iters += 1;

    // compute ritz values of p(A)
    cucheblanczos_ritz(ccm,ccl);

    // exit if converged
    if (ccl->nconv > neig) { 
      break; 
    }

  }

  // compute rayleigh quotients
  cucheblanczos_rayleigh(ccm,ccl);

  // sort evals
  cucheblanczos_sort(rho,ccl);

  // num_conv
  ccstats->num_conv = ccl->nconv;

  // max_res
  for(int ii=0; ii < ccl->nconv; ii++){
    ccstats->max_res = max(ccstats->max_res,ccl->res[ccl->index[ii]]);
  }

  // record compute time
  stop = time(0);
  ccstats->arnoldi_time = difftime(stop,start);

  // destroy ccp
  cuchebpoly_destroy(&ccp);

  // return  
  return 0;

}




/* expert lanczos routine for interval with statistics */
int cuchebmatrix_expertlanczos(double lbnd, double ubnd, int degree,
                                 int bsize, int numvecs, int stepsize, 
                                 cuchebmatrix* ccm, cucheblanczos* ccl, 
                                 cuchebstats* ccstats){

  // initialize ccstats
  ccstats->mat_dim = 0;
  ccstats->mat_nnz = 0;
  ccstats->block_size = 0;
  ccstats->num_blocks = 0;
  ccstats->num_iters = 0;
  ccstats->num_innerprods = 0;
  ccstats->max_degree = 0;
  ccstats->num_matvecs = 0;
  ccstats->specint_time = 0.0;
  ccstats->arnoldi_time = 0.0;
  ccstats->num_conv = 0;
  ccstats->max_res = 0.0;

  // collect some matrix statistics
  ccstats->mat_dim = ccm->m;
  ccstats->mat_nnz = ccm->nnz;

  // timing variables
  time_t start, stop;

  // compute spectral interval
  start = time(0);
  cuchebmatrix_specint(ccm);

  // record compute time
  stop = time(0);
  ccstats->specint_time = difftime(stop,start);

  // make sure lbnd is valid
  if (isnan(lbnd)) {
    printf("lbnd cannot be NaN!\n");
    exit(1);
  }

  // compute lower bound 
  double a, b;
  a = ccm->a;
  b = ccm->b;
  double lb;
  lb = min(max(a,lbnd),b);

  // compute upper bound 
  double ub;
  ub = max(min(b,ubnd),a);

  // make sure ubnd is valid
  if (lb >= ub) {
    printf("\ncuchebmatrix_filteredlanczos:\n");
    printf(" lb must be less than ub!\n\n");
    exit(1);
  }

  // initialize filter polynomial
  cuchebpoly ccp;
  cuchebpoly_init(&ccp);

  // create filter polynomial
  if (degree > -1) {
    cuchebpoly_stepfilter(ccm->a,ccm->b,lb,ub,degree,&ccp);
  }
  else {
    cuchebpoly_smartfilter(ccm->a,ccm->b,lb,ub,&ccp);
  }

  // max_degree
  ccstats->max_degree = max(ccstats->max_degree,ccp.degree);
    
  // initialize lanczos object
  cucheblanczos_init(bsize,numvecs,ccm,ccl);

  // collect some lanczos statistics
  ccstats->block_size = ccl->bsize;

  // set starting vector
  cucheblanczos_startvecs(ccl);

  // start stop watch
  start = time(0);

  // loop through various Krylov subspaces
  int numint = 0;
  int nres;
  int step;
  step = min(max(stepsize,1),ccl->nblocks);
  nres = (ccl->nblocks)/(step) + 1;
  for (int jj=0; jj<nres; jj++) {

    // filtered arnoldi run
    cucheblanczos_filteredarnoldi(step,ccm,&ccp,ccl,ccstats);

    // update ccstats
    // num_iters
    ccstats->num_iters += 1;

    // compute ritz values of p(A)
    cucheblanczos_ritz(ccm,ccl);

//  for(int ii=0; ii < ccl->nconv; ii++){
//    printf(" e,r = %+e,%e\n",ccl->evals[ccl->index[ii]],ccl->res[ccl->index[ii]]);
//  }
//printf("\n");

    // check to see if in interval
    numint = 0;
    for(int ii=0; ii<ccl->nconv; ii++){
      if(ccl->evals[ccl->index[ii]] >= .49){ numint += 1; }
      else { break; }
    }

    // exit if converged
    if (ccl->nconv > numint) { 
      ccl->nconv = numint;
      break; 
    }

  }

  // compute rayleigh quotients
  cucheblanczos_rayleigh(ccm,ccl);

  // sort evals
  cucheblanczos_sort(lb,ub,ccl);

  // num_conv
  ccstats->num_conv = ccl->nconv;

  // max_res
  for(int ii=0; ii < ccl->nconv; ii++){
    ccstats->max_res = max(ccstats->max_res,ccl->res[ccl->index[ii]]);
  }

  // record compute time
  stop = time(0);
  ccstats->arnoldi_time = difftime(stop,start);

  // destroy ccp
  cuchebpoly_destroy(&ccp);

  // return  
  return 0;

}
