#include "hip/hip_runtime.h"
#include <cucheb.h>

/* filtered lanczos routine */
int cuchebmatrix_filteredlanczos(int neig, double shift, cuchebmatrix* ccm,
                                 cucheblanczos* ccl){

  // compute spectral interval
  cuchebmatrix_specint(ccm);
  cuchebmatrix_print(ccm);

  // create filter polynomial
  cuchebpoly ccp;
  cuchebpoly_init(&ccp);
  cuchebpoly_pointfilter(ccm->a,ccm->b,shift,100*(ccm->m),&ccp);
  cuchebpoly_print(&ccp);

  // number of arnoldi steps
  int nvecs;
  nvecs = min(ccm->m,200);

  // initialize lanczos object
  cucheblanczos_init(nvecs,ccm,ccl);

  // set starting vector
  cucheblanczos_startvec(ccl);

  // filtered arnoldi run
  cucheblanczos_filteredarnoldi(ccm,&ccp,ccl);

  // compute ritz values
  cucheblanczos_eig(ccm,ccl);

  // print eigenvalues
  for(int ii=0; ii < ccl->nvecs; ii++){
    printf(" diag[%d] = %+e, sdiag[%d] = %+e\n",
           ii,ccl->diag[ii],ii,ccl->sdiag[ii]);
  }
  printf("\n");

  // compute rayleigh quotients
  cucheblanczos_rayleigh(ccm,ccl);

  // print eigenvalues
  for(int ii=0; ii < ccl->nvecs; ii++){
    printf(" diag[%d] = %+e, sdiag[%d] = %+e\n",
           ii,ccl->diag[ii],ii,ccl->sdiag[ii]);
  }
  printf("\n");

  // destroy ccp
  cuchebpoly_destroy(&ccp);

  // return  
  return 0;

}
