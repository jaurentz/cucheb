/*-----------------------------------------------------------------



-----------------------------------------------------------------*/

#include <cucheb.h>

/* constructors */
/* default */
ChebPoly::ChebPoly(void){

	// compute variables
	float temp;
	
	// set field
	field = CUCHEB_FIELD_FLOAT;
	
	// set degree
	degree = 0;
	
	// set a and b
	cuchebCheckError(hipMalloc(&a, sizeof(float)),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&b, sizeof(float)),__FILE__,__LINE__);
	temp = -1.0f;
	cuchebCheckError(hipMemcpy(a, &temp, sizeof(float), hipMemcpyHostToDevice),__FILE__,__LINE__);
	temp = 1.0f;
	cuchebCheckError(hipMemcpy(b, &temp, sizeof(float), hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// set coeffs
	cuchebCheckError(hipMalloc(&coeffs, sizeof(float)),__FILE__,__LINE__);
	temp = 1.0f;
	cuchebCheckError(hipMemcpy(coeffs, &temp, sizeof(float), hipMemcpyHostToDevice),__FILE__,__LINE__);
}

/* identity */
ChebPoly::ChebPoly(cuchebField_t F){

	// compute variables
	float stemp;
	double dtemp;
	hipComplex ctemp;
	hipDoubleComplex ztemp;
	
	// set field
	if(F == CUCHEB_FIELD_FLOAT){field = CUCHEB_FIELD_FLOAT;}
	else if(F == CUCHEB_FIELD_DOUBLE){field = CUCHEB_FIELD_DOUBLE;}
	else if(F == CUCHEB_FIELD_FLOAT_COMPLEX){field = CUCHEB_FIELD_FLOAT_COMPLEX;}
	else if(F == CUCHEB_FIELD_DOUBLE_COMPLEX){field = CUCHEB_FIELD_DOUBLE_COMPLEX;}
	else{printf("\nWarning in Chebpoly: Not a valid input for field! Setting field to CUCHEB_FIELD_FLOAT.\n\n");}
	
	// set a and b
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMalloc(&a, sizeof(float)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(float)),__FILE__,__LINE__);
		stemp = -1.0f;
		cuchebCheckError(hipMemcpy(a, &stemp, sizeof(float), hipMemcpyHostToDevice),__FILE__,__LINE__);
		stemp = 1.0f;
		cuchebCheckError(hipMemcpy(b, &stemp, sizeof(float), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMalloc(&a, sizeof(double)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(double)),__FILE__,__LINE__);
		dtemp = -1.0;
		cuchebCheckError(hipMemcpy(a, &dtemp, sizeof(double), hipMemcpyHostToDevice),__FILE__,__LINE__);
		dtemp = 1.0;
		cuchebCheckError(hipMemcpy(b, &dtemp, sizeof(double), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMalloc(&a, sizeof(hipComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(hipComplex)),__FILE__,__LINE__);
		ctemp = make_hipFloatComplex(-1.0f,0.0f);
		cuchebCheckError(hipMemcpy(a, &ctemp, sizeof(hipComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
		ctemp = make_hipFloatComplex(1.0f,0.0f);
		cuchebCheckError(hipMemcpy(b, &ctemp, sizeof(hipComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMalloc(&a, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		ztemp = make_hipDoubleComplex(-1.0,0.0);
		cuchebCheckError(hipMemcpy(a, &ztemp, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
		ztemp = make_hipDoubleComplex(1.0,0.0);
		cuchebCheckError(hipMemcpy(b, &ztemp, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}
	
	// set coeffs
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMalloc(&coeffs, sizeof(float)),__FILE__,__LINE__);
		stemp = 1.0f;
		cuchebCheckError(hipMemcpy(coeffs, &stemp, sizeof(float), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMalloc(&coeffs, sizeof(double)),__FILE__,__LINE__);
		dtemp = 1.0;
		cuchebCheckError(hipMemcpy(coeffs, &dtemp, sizeof(double), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMalloc(&coeffs, sizeof(hipComplex)),__FILE__,__LINE__);
		ctemp = make_hipFloatComplex(1.0f,0.0f);
		cuchebCheckError(hipMemcpy(coeffs, &ctemp, sizeof(hipComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMalloc(&coeffs, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		ztemp = make_hipDoubleComplex(1.0,0.0);
		cuchebCheckError(hipMemcpy(coeffs, &ztemp, sizeof(hipDoubleComplex), hipMemcpyHostToDevice),__FILE__,__LINE__);
	}	
}

/* copy */
ChebPoly::ChebPoly(const ChebPoly& CP){

	// compute variables
	void *temp;

	// set field
	field = CP.getField();
	
	// set degree
	degree = CP.getDegree();
	
	// set a and b
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMalloc(&a, sizeof(float)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(float)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(float), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(float), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMalloc(&a, sizeof(double)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(double)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(double), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(double), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMalloc(&a, sizeof(hipComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(hipComplex)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(hipComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(hipComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMalloc(&a, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	
	// set coeffs
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(float)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(float), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(double)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(double), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(hipComplex)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(hipComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
}

/* destructor */
ChebPoly::~ChebPoly(void){

	// free memory
	cuchebCheckError(hipFree(a),__FILE__,__LINE__);
	cuchebCheckError(hipFree(b),__FILE__,__LINE__);
	cuchebCheckError(hipFree(coeffs),__FILE__,__LINE__);
}

/* printers */
/* print short */
void ChebPoly::print(void){

	// compute variables
	float sa, sb;
	double da, db;
	hipComplex ca, cb;
	hipDoubleComplex za, zb;

	// header
	printf("\nChebPoly:\n");

	// field
	if(field == CUCHEB_FIELD_FLOAT){printf(" field = CUCHEB_FIELD_FLOAT\n");}
	else if(field == CUCHEB_FIELD_DOUBLE){printf(" field = CUCHEB_FIELD_DOUBLE\n");}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){printf(" field = CUCHEB_FIELD_FLOAT_COMPLEX\n");}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){printf(" field = CUCHEB_FIELD_DOUBLE_COMPLEX\n");}	

	// degree
	printf(" degree = %d\n",degree);
	
	// a and b
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMemcpy(&sa, a, sizeof(float), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&sb, b, sizeof(float), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [%+e,%+e]\n",sa,sb);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMemcpy(&da, a, sizeof(double), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&db, b, sizeof(double), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [%+1.15e,%+1.15e]\n",da,db);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMemcpy(&ca, a, sizeof(hipComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&cb, b, sizeof(hipComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [(%+e,%+e),(%+e,%+e)]\n",hipCrealf(ca),hipCimagf(ca),hipCrealf(cb),hipCimagf(cb));
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMemcpy(&za, a, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&zb, b, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [(%+1.15e,%+1.15e),(%+1.15e,%+1.15e)]\n",hipCreal(za),hipCimag(za),hipCreal(zb),hipCimag(zb));
	}

	// footer
	printf("\n");
}

/* print long */
void ChebPoly::printlong(void){

	// compute variables
	float sa, sb, *scoeffs;
	double da, db, *dcoeffs;
	hipComplex ca, cb, *ccoeffs;
	hipDoubleComplex za, zb, *zcoeffs;

	// header
	printf("\nChebPoly:\n");

	// field
	if(field == CUCHEB_FIELD_FLOAT){printf(" field = CUCHEB_FIELD_FLOAT\n");}
	else if(field == CUCHEB_FIELD_DOUBLE){printf(" field = CUCHEB_FIELD_DOUBLE\n");}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){printf(" field = CUCHEB_FIELD_FLOAT_COMPLEX\n");}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){printf(" field = CUCHEB_FIELD_DOUBLE_COMPLEX\n");}	

	// degree
	printf(" degree = %d\n",degree);
	
	// a and b
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMemcpy(&sa, a, sizeof(float), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&sb, b, sizeof(float), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [%+e,%+e]\n",sa,sb);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMemcpy(&da, a, sizeof(double), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&db, b, sizeof(double), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [%+1.15e,%+1.15e]\n",da,db);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMemcpy(&ca, a, sizeof(hipComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&cb, b, sizeof(hipComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [(%+e,%+e),(%+e,%+e)]\n",hipCrealf(ca),hipCimagf(ca),hipCrealf(cb),hipCimagf(cb));
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMemcpy(&za, a, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(&zb, b, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		printf(" [a,b] = [(%+1.15e,%+1.15e),(%+1.15e,%+1.15e)]\n",hipCreal(za),hipCimag(za),hipCreal(zb),hipCimag(zb));
	}
	
	// coeffs
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError((void*)(scoeffs = (float*)malloc((degree+1)*sizeof(float))),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(scoeffs, coeffs, (degree+1)*sizeof(float), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		for(int ii=0;ii<(degree+1);ii++){printf(" coeffs[%d] = %+e\n",ii,scoeffs[ii]);}
		free(scoeffs);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError((void*)(dcoeffs = (double*)malloc((degree+1)*sizeof(double))),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(dcoeffs, coeffs, (degree+1)*sizeof(double), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		for(int ii=0;ii<(degree+1);ii++){printf(" coeffs[%d] = %+1.15e\n",ii,dcoeffs[ii]);}
		free(dcoeffs);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError((void*)(ccoeffs = (hipComplex*)malloc((degree+1)*sizeof(hipComplex))),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(ccoeffs, coeffs, (degree+1)*sizeof(hipComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		for(int ii=0;ii<(degree+1);ii++){printf(" coeffs[%d] = (%+e,%+e)\n",ii,hipCrealf(ccoeffs[ii]),hipCimagf(ccoeffs[ii]));}
		free(ccoeffs);
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError((void*)(zcoeffs = (hipDoubleComplex*)malloc((degree+1)*sizeof(hipDoubleComplex))),__FILE__,__LINE__);
		cuchebCheckError(hipMemcpy(zcoeffs, coeffs, (degree+1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost),__FILE__,__LINE__);
		for(int ii=0;ii<(degree+1);ii++){printf(" coeffs[%d] = (%+1.15e,%+1.15e)\n",ii,hipCreal(zcoeffs[ii]),hipCimag(zcoeffs[ii]));}
		free(zcoeffs);
	}

	// footer
	printf("\n");
}

// operator overload
ChebPoly& ChebPoly::operator= (const ChebPoly& CP){

	// check for self-assignment
    if(this == &CP){return *this;}

	// compute variables
	void *temp;

	// set field
	field = CP.getField();
	
	// set degree
	degree = CP.getDegree();
	
	// set a and b
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMalloc(&a, sizeof(float)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(float)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(float), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(float), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMalloc(&a, sizeof(double)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(double)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(double), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(double), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMalloc(&a, sizeof(hipComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(hipComplex)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(hipComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(hipComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMalloc(&a, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		cuchebCheckError(hipMalloc(&b, sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		temp = CP.getA();
		cuchebCheckError(hipMemcpy(a, temp, sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
		temp = CP.getB();
		cuchebCheckError(hipMemcpy(b, temp, sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	
	// set coeffs
	if(field == CUCHEB_FIELD_FLOAT){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(float)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(float), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(double)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(double), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_FLOAT_COMPLEX){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(hipComplex)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(hipComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	else if(field == CUCHEB_FIELD_DOUBLE_COMPLEX){
		cuchebCheckError(hipMalloc(&coeffs, (degree+1)*sizeof(hipDoubleComplex)),__FILE__,__LINE__);
		temp = CP.getCoeffs();
		cuchebCheckError(hipMemcpy(coeffs, temp, (degree+1)*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice),__FILE__,__LINE__);
	}
	
	// return
	return *this;
}

