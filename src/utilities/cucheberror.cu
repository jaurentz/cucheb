/*-----------------------------------------------------------------



-----------------------------------------------------------------*/

#include <cucheb.h>

/* cucheb exit */
void cuchebExit(int ii){
	hipDeviceReset();
	exit(ii);
}

/* cucheb error string */
static const char* cuchebGetErrorString(cuchebStatus_t err){

	switch(err){
	    case CUCHEB_STATUS_SUCCESS:
	       return "CUCHEB_STATUS_SUCCESS"; 
	    case CUCHEB_STATUS_CUDA_FAILED:
	       return "CUCHEB_STATUS_CUDA_FAILED";
	    case CUCHEB_STATUS_CURAND_FAILED:
	       return "CUCHEB_STATUS_CURAND_FAILED"; 
	    case CUCHEB_STATUS_CUFFT_FAILED:
	       return "CUCHEB_STATUS_CUFFT_FAILED";
	    case CUCHEB_STATUS_CUBLAS_FAILED:
	       return "CUCHEB_STATUS_CUBLAS_FAILED"; 
	    case CUCHEB_STATUS_CUSPARSE_FAILED:
	       return "CUCHEB_STATUS_CUSPARSE_FAILED";
	    default: 
	       return "CUCHEB_STATUS_UNKNOWN";
	}
}

/* cublas error string */
static const char* cublasGetErrorString(hipblasStatus_t err){

    switch (err){
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
        default: 
	    	return "CUBLAS_STATUS_UNKNOWN";
    }
}

/* hiprand error string */
static const char* curandGetErrorString(hiprandStatus_t err)
{
    switch (err){
        case HIPRAND_STATUS_SUCCESS:
            return "HIPRAND_STATUS_SUCCESS";
        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";
        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";
        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";
        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";
        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
        case HIPRAND_STATUS_LAUNCH_FAILURE:
            return "HIPRAND_STATUS_LAUNCH_FAILURE";
        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";
        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";
        case HIPRAND_STATUS_ARCH_MISMATCH:
            return "HIPRAND_STATUS_ARCH_MISMATCH";
        case HIPRAND_STATUS_INTERNAL_ERROR:
            return "HIPRAND_STATUS_INTERNAL_ERROR";
        default:
            return "CURAND_STATUS_UNKNOWN";
    }
}

/* cufft error string */
static const char* cufftGetErrorString(hipfftResult err)
{
    switch (err){
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";
        case HIPFFT_INCOMPLETE_PARAMETER_LIST:
            return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
        case HIPFFT_INVALID_DEVICE:
            return "HIPFFT_INVALID_DEVICE";
        case HIPFFT_PARSE_ERROR:
            return "HIPFFT_PARSE_ERROR";
        case HIPFFT_NO_WORKSPACE:
            return "HIPFFT_NO_WORKSPACE";
        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";
        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
        default: 
            return "CUFFT_UNKNOWN";
	}
}

/* cusparse error string */
static const char* hipsparseGetErrorString(hipsparseStatus_t err)
{
    switch (err){
        case HIPSPARSE_STATUS_SUCCESS:
            return "HIPSPARSE_STATUS_SUCCESS";
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "HIPSPARSE_STATUS_NOT_INITIALIZED";
        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "HIPSPARSE_STATUS_ALLOC_FAILED";
        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "HIPSPARSE_STATUS_INVALID_VALUE";
        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "HIPSPARSE_STATUS_ARCH_MISMATCH";
        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "HIPSPARSE_STATUS_MAPPING_ERROR";
        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "HIPSPARSE_STATUS_EXECUTION_FAILED";
        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "HIPSPARSE_STATUS_INTERNAL_ERROR";
        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        default: 
            return "CUSPARSE_STATUS_UNKNOWN";
	}
}

/* host pointer */
void cuchebCheckError(void* err,char* file,int line){

	/* print error */
	if(err == NULL){
		fprintf(stderr,"\nHost memory allocation failure occured in %s at line: %d\n\n",file,line);
		cuchebExit(-1);
	}
} 

/* cucheb */
void cuchebCheckError(cuchebStatus_t err,char* file,int line){

	/* print error */
	if(err != CUCHEB_STATUS_SUCCESS){
		fprintf(stderr,"\n%s occured in %s at line: %d\n\n",cuchebGetErrorString(err),file,line);
		cuchebExit(-1);
	}
}  

/* cuda */
void cuchebCheckError(hipError_t err,char* file,int line){

	/* print error and return CUDA_FAILED */
	if(err != hipSuccess){
		fprintf(stderr,"\nCUCHEB_STATUS_CUDA_FAILED occured in %s at line: %d\n    cuda error: %s\n\n",
			file,line,hipGetErrorString(err));
		cuchebExit(-1);
	}
} 

/* cublas */
void cuchebCheckError(hipblasStatus_t err,char* file,int line){

	/* print error and return CUBLAS_FAILED */
	if(err != HIPBLAS_STATUS_SUCCESS){
		fprintf(stderr,"\nCUCHEB_STATUS_CUBLAS_FAILED occured in %s at line: %d\n    cublas error: %s\n\n",
			file,line,cublasGetErrorString(err));
		cuchebExit(-1);
	}
}       

/* hiprand */
void cuchebCheckError(hiprandStatus_t err,char* file,int line){

	/* print error and return CURAND_FAILED */
	if(err != HIPRAND_STATUS_SUCCESS){
		fprintf(stderr,"\nCUCHEB_STATUS_CURAND_FAILED occured in %s at line: %d\n    hiprand error: %s\n\n",
			file,line,curandGetErrorString(err));
		cuchebExit(-1);
	}
} 

/* cufft */
void cuchebCheckError(hipfftResult err,char* file,int line){

	/* print error and return CUFFT_FAILED */
	if(err != HIPFFT_SUCCESS){
		fprintf(stderr,"\nCUCHEB_STATUS_CUFFT_FAILED occured in %s at line: %d\n    cufft error: %s\n\n",
			file,line,cufftGetErrorString(err));
		cuchebExit(-1);
	}
} 

/* cusparse */
void cuchebCheckError(hipsparseStatus_t err,char* file,int line){

	/* print error and return CUSPARSE_FAILED */
	if(err != HIPSPARSE_STATUS_SUCCESS){
		fprintf(stderr,"\nCUCHEB_STATUS_CUSPARSE_FAILED occured in %s at line: %d\n    cusparse error: %s\n\n",
			file,line,hipsparseGetErrorString(err));
		cuchebExit(-1);
	}
} 
