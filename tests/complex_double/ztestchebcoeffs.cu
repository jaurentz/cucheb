#include "hip/hip_runtime.h"
#include <cucheb.h>

// prototypes for sample functions to interpolate
/* complex double precision */
__host__ __device__ void ztestfun(const hipDoubleComplex *x, hipDoubleComplex *y);
__global__ void ztestkernel(int n, const hipDoubleComplex *in, int incin, hipDoubleComplex* out, int incout);
cuchebStatus_t ztestcaller(int n, const hipDoubleComplex *in, int incin, hipDoubleComplex* out, int incout);

// driver
int main(){

	// compute variables
	/* complex double */
	int zn = pow(2,16)+1;
	int sizeZ = sizeof(hipDoubleComplex);
	hipDoubleComplex *zpts, *za, *zb, *zcfs, *zfvs, *dzpts, *dza, *dzb, *dzcfs, *dzfvs;
	
	// allocate host memory
	/* complex double */
	cuchebCheckError((void*)(zpts = (hipDoubleComplex*)malloc(zn*sizeZ)),__FILE__,__LINE__);
	cuchebCheckError((void*)(zcfs = (hipDoubleComplex*)malloc(zn*sizeZ)),__FILE__,__LINE__);
	cuchebCheckError((void*)(zfvs = (hipDoubleComplex*)malloc(zn*sizeZ)),__FILE__,__LINE__);
	cuchebCheckError((void*)(za = (hipDoubleComplex*)malloc(sizeZ)),__FILE__,__LINE__);
	cuchebCheckError((void*)(zb = (hipDoubleComplex*)malloc(sizeZ)),__FILE__,__LINE__);
	
	// allocate device memory
	/* complex double */
	cuchebCheckError(hipMalloc(&dzpts, zn*sizeZ),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dzcfs, zn*sizeZ),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dzfvs, zn*sizeZ),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dza, sizeZ),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dzb, sizeZ),__FILE__,__LINE__);
	
	// set host pointers
	/* complex double */
	*za = make_hipDoubleComplex(-1.0,-1.0);
	*zb = make_hipDoubleComplex(1.0,3.0);
	
	// copy host memory to device memory
	/* complex double */
	cuchebCheckError(hipMemcpy(dza, za, sizeZ, hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(dzb, zb, sizeZ, hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// compute chebpoints
	/* complex double */
	cuchebCheckError(cuchebZpoints(zn,dza,dzb,dzpts,1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex double */
	cuchebCheckError(hipMemcpy(zpts, dzpts, zn*sizeZ, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// compute funvals
	/* complex double */
	cuchebCheckError(ztestcaller(zn, dzpts, 1, dzfvs, 1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex double */
	cuchebCheckError(hipMemcpy(zfvs, dzfvs, zn*sizeZ, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// compute chebcoeffs
	/* complex double */
	cuchebCheckError(cuchebZcoeffs(zn, dzfvs, 1, dzcfs, 1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex double */
	cuchebCheckError(hipMemcpy(zcfs, dzcfs, zn*sizeZ, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// print output
	/* complex double */
	printf("complex double precision\n");
	for(int ii=0;ii<8;ii++){printf("zfvs[%d] = (%+1.15e,%+1.15e), zcfs[%d] = (%+1.15e,%+1.15e)\n",
		ii,hipCreal(zfvs[ii]),hipCimag(zfvs[ii]),ii,hipCreal(zcfs[ii]),hipCimag(zcfs[ii]));}
	printf("\n");

	// free device memory
	/* complex double */
	cuchebCheckError(hipFree(dzpts),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dzcfs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dzfvs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dza),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dzb),__FILE__,__LINE__);

	// free host memory
	/* complex double */
	free(zpts);
	free(zcfs);
	free(zfvs);
	free(za);
	free(zb);
	
	return 0;
}

// sample functions to interpolate
/* complex double precision */
__host__ __device__ void ztestfun(const hipDoubleComplex *x, hipDoubleComplex *y){
	//hipDoubleComplex temp;
	//hipDoubleComplex I = make_hipDoubleComplex(1.0,1.0);
	//temp = hipCmul(*x,I);
	*y = *x;
}
__global__ void ztestkernel(int n, const hipDoubleComplex *in, int incin, hipDoubleComplex* out, int incout){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;

	if(ii < n){
		ztestfun(&in[ii*incin],&out[ii*incout]);
	}
}
cuchebStatus_t ztestcaller(int n, const hipDoubleComplex *in, int incin, hipDoubleComplex* out, int incout){
	// check n
	if(n <= 0){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check incin
	if(incin <= 0){
		fprintf(stderr,"\nIn %s line: %d, incin must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check incout
	if(incout <= 0){
		fprintf(stderr,"\nIn %s line: %d, incout must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
		
	// query device
	int dev;
	hipDeviceProp_t prop;
	cuchebCheckError(hipGetDevice(&dev),__FILE__,__LINE__);
	cuchebCheckError(hipGetDeviceProperties(&prop,dev),__FILE__,__LINE__);
	
	// set blockSize
	int blockSize;
	blockSize = prop.maxThreadsPerBlock;
	
	// set gridSize
	int gridSize;
	gridSize = (int)ceil((double)n/blockSize);
	
	// launch fill input kernel
	ztestkernel<<<gridSize,blockSize>>>(n, in, incin, out, incout);
	
	// check for kernel error
	cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
	// return success
	return CUCHEB_STATUS_SUCCESS;
}
