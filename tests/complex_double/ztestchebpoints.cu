#include "hip/hip_runtime.h"
#include <cucheb.h>

int main(){

	// compute variables
	/* complex double */
	int zn = pow(2,3)+1;
	int sizeZ = sizeof(hipDoubleComplex);
	hipDoubleComplex *zpts, *za, *zb, *dzpts, *dza, *dzb;
	
	// allocate host memory
	/* complex double */
	cuchebCheckError((void*)(zpts = (hipDoubleComplex*)malloc(zn*sizeZ)),__FILE__,__LINE__);
	cuchebCheckError((void*)(za = (hipDoubleComplex*)malloc(sizeZ)),__FILE__,__LINE__);
	cuchebCheckError((void*)(zb = (hipDoubleComplex*)malloc(sizeZ)),__FILE__,__LINE__);
	
	// allocate device memory
	/* complex double */
	cuchebCheckError(hipMalloc(&dzpts, zn*sizeZ),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dza, sizeZ),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dzb, sizeZ),__FILE__,__LINE__);
	
	// set host pointers
	/* complex double */
	*za = make_hipDoubleComplex(-1.0,0.0);
	*zb = make_hipDoubleComplex(1.0,0.0);
	
	// copy host memory to device memory
	/* complex double */
	cuchebCheckError(hipMemcpy(dza, za, sizeZ, hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(dzb, zb, sizeZ, hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// compute chebpoints
	/* complex double */
	cuchebCheckError(cuchebZpoints(zn,dza,dzb,dzpts,1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex double */
	cuchebCheckError(hipMemcpy(zpts, dzpts, zn*sizeZ, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// print output
	/* complex double */
	printf("complex double precision\n");
	for(int ii=0;ii<zn;ii++){printf("zpts[%d] = (%+1.15e,%+1.15e)\n",ii,hipCreal(zpts[ii]),hipCimag(zpts[ii]));}
	printf("\n");

	// free device memory
	/* complex double */
	cuchebCheckError(hipFree(dzpts),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dza),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dzb),__FILE__,__LINE__);

	// free host memory
	/* complex double */
	free(zpts);
	free(za);
	free(zb);
	
	return 0;
}
