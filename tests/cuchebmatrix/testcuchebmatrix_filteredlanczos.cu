#include <cucheb.h>

/* driver */
int main(){

  // set device
  hipSetDevice(1);

  // cuhebmatrix
  string mtxfile("../matrices/ca2010.mtx");
  cuchebmatrix ccm;
  cuchebmatrix_init(mtxfile, &ccm);

  // call filtered lanczos for an interval
  cucheblanczos ccl;
  cuchebstats ccstats;
  //cuchebmatrix_filteredlanczos(4.0e6, 4.1e6, 1, &ccm, &ccl, &ccstats);
  cuchebmatrix_expertlanczos(-1.0e4, 1.0e4, -1, 1, 1000, 50, &ccm, &ccl, &ccstats);

  // print ccm
  cuchebmatrix_print(&ccm);

  // print ccstats
  cuchebstats_print(&ccstats);

  // print eigenvalues
  for (int ii=0; ii<ccl.nconv; ii++) {
  //for (int ii=0; ii<100; ii++) {
    printf(" %+e, %e\n",ccl.evals[ccl.index[ii]],ccl.res[ccl.index[ii]]);
  }
  printf("\n");

  // destroy CCM
  cuchebmatrix_destroy(&ccm);

  // destroy CCB
  cucheblanczos_destroy(&ccl);

  // return 
  return 0;

}
