#include <cucheb.h>

/* driver */
int main(){

  // set device
  //hipSetDevice(1);

  // input file
  //string mtxfile("../matrices/SiH4.mtx");
  //string mtxfile("../matrices/Si10H16.mtx");
  //string mtxfile("../matrices/H2O.mtx");
  //string mtxfile("../matrices/Si34H36.mtx");
  //string mtxfile("../matrices/Si87H76.mtx");
  //string mtxfile("../matrices/CO.mtx");
  //string mtxfile("../matrices/Ga41As41H72.mtx");
  //string mtxfile("../matrices/Ge99H100.mtx");
  //string mtxfile("../matrices/Andrews.mtx");
  //string mtxfile("../matrices/Laplacian.mtx");
  //string mtxfile("../matrices/Qdot3.mtx");
  //string mtxfile("../matrices/DIMACS/144.mtx");
  //string mtxfile("../matrices/DIMACS/598a.mtx");
  string mtxfile("../matrices/DIMACS/al2010.mtx");
  //string mtxfile("../matrices/DIMACS/ar2010.mtx");
  //string mtxfile("../matrices/DIMACS/auto.mtx");
  //string mtxfile("../matrices/DIMACS/az2010.mtx");
  //string mtxfile("../matrices/DIMACS/ca2010.mtx");

  // cuhebmatrix
  cuchebmatrix ccm;
  cuchebmatrix_init(mtxfile, &ccm);

  // cucheblanczos
  cucheblanczos ccl;

  // cuchebstats
  cuchebstats ccstats;

  //cuchebmatrix_specint(&ccm);

  // call filtered lanczos for a point
  //cuchebmatrix_filteredlanczos(100, 1.0e300, 1, &ccm, &ccl, &ccstats);

  // call filtered lanczos for an interval
  //cuchebmatrix_filteredlanczos(-0.66, -.33, 3, &ccm, &ccl, &ccstats);

  // call expert lanczos
  //cuchebmatrix_expertlanczos(4.00, 5.00, 150, 1, 4000, 4000, &ccm, &ccl, &ccstats);
  //cuchebmatrix_expertlanczos(1.00, 1.01, 1600, 3, 1200, 400, &ccm, &ccl, &ccstats);
  //cuchebmatrix_expertlanczos(-2.0, -.33, 100, 3, 1200, 40, &ccm, &ccl, &ccstats);
  cuchebmatrix_expertlanczos(1.0e6, 1.0e100, -1, 1, 3000, 500, &ccm, &ccl, &ccstats);

  // print ccm
  cuchebmatrix_print(&ccm);

  // print ccstats
  cuchebstats_print(&ccstats);

  // print eigenvalues
  for (int ii=0; ii<ccl.nconv; ii++) {
  //for (int ii=0; ii<100; ii++) {
    printf(" %+e, %e\n",ccl.evals[ccl.index[ii]],ccl.res[ccl.index[ii]]);
  }
  printf("\n");

  // destroy CCM
  cuchebmatrix_destroy(&ccm);

  // destroy CCB
  cucheblanczos_destroy(&ccl);

  // return 
  return 0;

}
