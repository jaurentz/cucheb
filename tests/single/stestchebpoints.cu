#include "hip/hip_runtime.h"
#include <cucheb.h>

int main(){

	// compute variables
	/* single */
	int sn = pow(2,3)+1;
	int sizeS = sizeof(float);
	float *spts, *sa, *sb, *dspts, *dsa, *dsb;
	
	// allocate host memory
	/* single */
	cuchebCheckError((void*)(spts = (float*)malloc(sn*sizeS)),__FILE__,__LINE__);
	cuchebCheckError((void*)(sa = (float*)malloc(sizeS)),__FILE__,__LINE__);
	cuchebCheckError((void*)(sb = (float*)malloc(sizeS)),__FILE__,__LINE__);
	
	// allocate device memory
	/* single */
	cuchebCheckError(hipMalloc(&dspts, sn*sizeS),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dsa, sizeS),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dsb, sizeS),__FILE__,__LINE__);
	
	// set host pointers
	/* single */
	*sa = -1.0f;
	*sb = 1.0f;
	
	// copy host memory to device memory
	/* single */
	cuchebCheckError(hipMemcpy(dsa, sa, sizeS, hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(dsb, sb, sizeS, hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// compute chebpoints
	/* single */
	cuchebCheckError(cuchebSpoints(sn,dsa,dsb,dspts,1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* single */
	cuchebCheckError(hipMemcpy(spts, dspts, sn*sizeS, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// print output
	/* single */
	printf("single precision\n");
	for(int ii=0;ii<sn;ii++){printf("spts[%d] = %+e\n",ii,spts[ii]);}
	printf("\n");

	// free device memory
	/* single */
	cuchebCheckError(hipFree(dspts),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dsa),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dsb),__FILE__,__LINE__);

	// free host memory
	/* single */
	free(spts);
	free(sa);
	free(sb);
	
	return 0;
}
