#include <cucheb.h>

/* driver */
int main(){

  // input file
  //string mtxfile("./matrices/Sandi_authors.mtx");
  //string mtxfile("../matrices/Trefethen_20.mtx");
  string mtxfile("../matrices/Stranke94.mtx");

  // cuchebmatrix
  cuchebmatrix ccm;
  cuchebmatrix_init(mtxfile, &ccm);

  // cucheblanczos
  cucheblanczos ccl;
  cucheblanczos_init(2, 5, &ccm, &ccl);

  // print CCB
  cucheblanczos_print(&ccl);

  // set starting vector
  cucheblanczos_startvecs(&ccl);

  // do arnoldi run
  cucheblanczos_arnoldi(&ccm,&ccl);

  // print arnoldi vectors
  double val;
  int nvecs;
  nvecs = (ccl.bsize)*(ccl.nblocks);
  for(int jj=0; jj < nvecs+ccl.bsize; jj++){
  for(int ii=0; ii < ccl.n; ii++){
    hipMemcpy(&val,&(ccl.dvecs)[jj*ccl.n + ii],sizeof(double),hipMemcpyDeviceToHost);
    printf(" dvecs[%d] = %+e\n", jj*ccl.n+ii, val);
  }
  printf("\n");
  }
  printf("\n");

  // compute ritz values
  cucheblanczos_eig(&ccm,&ccl);

  // print bands
  for(int ii=0; ii < nvecs; ii++){

    for(int jj=0; jj < nvecs+ccl.bsize; jj++){
      printf(" schurvecs[%d] = %+e\n", ii*(nvecs+ccl.bsize)+jj, ccl.schurvecs[ii*(nvecs+ccl.bsize)+jj]);
    }
    printf("\n");

    for(int jj=0; jj < ccl.bsize+1; jj++){
      printf(" bands[%d] = %+e\n", ii*(ccl.bsize+1)+jj,
             ccl.bands[ii*(ccl.bsize+1)+jj]);
    }
    printf("\n");

  }
  printf("\n");

  // print evals
  for(int ii=0; ii < nvecs; ii++){
    printf(" evals[%d] = %+e\n", ii, ccl.evals[ii]);
  }
  printf("\n");

  // print ritz vectors
  for(int jj=0; jj < nvecs; jj++){
  for(int ii=0; ii < ccl.n; ii++){
    hipMemcpy(&val,&(ccl.dvecs)[jj*ccl.n + ii],sizeof(double),hipMemcpyDeviceToHost);
    printf(" dvecs[%d] = %+e\n", jj*ccl.n+ii, val);
  }
  printf("\n");
  }
  printf("\n");

  // compute rayleigh quotients and residuals
  cucheblanczos_rayleigh(&ccm,&ccl);

  // print ritz vectors
  for(int jj=0; jj < nvecs; jj++){
    printf(" evals[%d] = %+e, res[%d] = %+e\n", jj, ccl.evals[jj], 
           jj, ccl.res[jj]);
  }
  printf("\n");

  // destroy CCM
  cuchebmatrix_destroy(&ccm);

  // destroy CCL
  cucheblanczos_destroy(&ccl);

  // return 
  return 0;

}
