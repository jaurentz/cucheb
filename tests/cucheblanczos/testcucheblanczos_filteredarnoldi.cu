#include <cucheb.h>

/* driver */
int main(){

  // input file
  string mtxfile("../matrices/Stranke94.mtx");

  // cuchebmatrix
  cuchebmatrix ccm;
  cuchebmatrix_init(mtxfile, &ccm);
  cuchebmatrix_specint(&ccm);
  cuchebmatrix_print(&ccm);

  // filter polynomial
  cuchebpoly ccp;
  cuchebpoly_init(&ccp);
  cuchebpoly_pointfilter(ccm.a,ccm.b,0,10,&ccp);
  cuchebpoly_print(&ccp);

  // cucheblanczos
  cucheblanczos ccl;
  cucheblanczos_init(1, MAX_NUM_VECS, &ccm, &ccl);

  // print CCB
  cucheblanczos_print(&ccl);

  // set starting vector
  cucheblanczos_startvecs(&ccl);

  // cuchebstats
  cuchebstats ccstats;

  // do arnoldi run
  cucheblanczos_filteredarnoldi(5,&ccm,&ccp,&ccl,&ccstats);

  // print ccl
  cucheblanczos_print(&ccl);

  // print arnoldi vectors
  double val;
  int nvecs;
  nvecs = (ccl.bsize)*(ccl.nblocks);
  for(int jj=0; jj < nvecs+ccl.bsize; jj++){
  for(int ii=0; ii < ccl.n; ii++){
    hipMemcpy(&val,&(ccl.dvecs)[jj*ccl.n + ii],sizeof(double),hipMemcpyDeviceToHost);
    printf(" dvecs[%d] = %+e\n", jj*ccl.n+ii, val);
  }
  printf("\n");
  }
  printf("\n");

  // compute ritz values
  cucheblanczos_ritz(&ccm,&ccl);

  // print bands
  for(int ii=0; ii < nvecs; ii++){

    for(int jj=0; jj < nvecs+ccl.bsize; jj++){
      printf(" schurvecs[%d] = %+e\n", ii*(nvecs+ccl.bsize)+jj, ccl.schurvecs[ii*(nvecs+ccl.bsize)+jj]);
    }
    printf("\n");

    for(int jj=0; jj < ccl.bsize+1; jj++){
      printf(" bands[%d] = %+e\n", ii*(ccl.bsize+1)+jj,
             ccl.bands[ii*(ccl.bsize+1)+jj]);
    }
    printf("\n");

  }
  printf("\n");

  // print evals
  for(int ii=0; ii < nvecs; ii++){
    printf(" evals[%d] = %+e\n", ii, ccl.evals[ii]);
  }
  printf("\n");

  // print ritz vectors
  for(int jj=0; jj < nvecs; jj++){
  for(int ii=0; ii < ccl.n; ii++){
    hipMemcpy(&val,&(ccl.dvecs)[jj*ccl.n + ii],sizeof(double),hipMemcpyDeviceToHost);
    printf(" dvecs[%d] = %+e\n", jj*ccl.n+ii, val);
  }
  printf("\n");
  }
  printf("\n");

  // compute rayleigh quotients and residuals
  cucheblanczos_rayleigh(&ccm,&ccl);

  // print ritz vectors
  for(int jj=0; jj < (ccl.bsize)*(ccl.nblocks); jj++){
    printf(" evals[%d] = %+e, res[%d] = %+e\n", jj, ccl.evals[jj], 
           jj, ccl.res[jj]);
  }
  printf("\n");

  // destroy CCP
  cuchebpoly_destroy(&ccp);

  // destroy CCM
  cuchebmatrix_destroy(&ccm);

  // destroy CCL
  cucheblanczos_destroy(&ccl);

  // return 
  return 0;

}
