#include "hip/hip_runtime.h"
#include <cucheb.h>

int main(){

	// compute variables
	/* complex single */
	int cn = pow(2,3)+1;
	int sizeC = sizeof(hipComplex);
	hipComplex *cpts, *ca, *cb, *dcpts, *dca, *dcb;
	
	// allocate host memory
	/* complex single */
	cuchebCheckError((void*)(cpts = (hipComplex*)malloc(cn*sizeC)),__FILE__,__LINE__);
	cuchebCheckError((void*)(ca = (hipComplex*)malloc(sizeC)),__FILE__,__LINE__);
	cuchebCheckError((void*)(cb = (hipComplex*)malloc(sizeC)),__FILE__,__LINE__);
	
	// allocate device memory
	/* complex single */
	cuchebCheckError(hipMalloc(&dcpts, cn*sizeC),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dca, sizeC),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dcb, sizeC),__FILE__,__LINE__);
	
	// set host pointers
	/* complex single */
	*ca = make_hipFloatComplex(-1.0f,0.0f);
	*cb = make_hipFloatComplex(1.0f,0.0f);
	
	// copy host memory to device memory
	/* complex single */
	cuchebCheckError(hipMemcpy(dca, ca, sizeC, hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(dcb, cb, sizeC, hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// compute chebpoints
	/* complex single */
	cuchebCheckError(cuchebCpoints(cn,dca,dcb,dcpts,1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex single */
	cuchebCheckError(hipMemcpy(cpts, dcpts, cn*sizeC, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// print output
	/* complex single */
	printf("complex single precision\n");
	for(int ii=0;ii<cn;ii++){printf("cpts[%d] = (%+e,%+e)\n",ii,hipCrealf(cpts[ii]),hipCimagf(cpts[ii]));}
	printf("\n");

	// free device memory
	/* complex single */
	cuchebCheckError(hipFree(dcpts),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dca),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dcb),__FILE__,__LINE__);
	
	// free host memory
	/* complex single */
	free(cpts);
	free(ca);
	free(cb);
	
	return 0;
}
