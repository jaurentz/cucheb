#include "hip/hip_runtime.h"
#include <cucheb.h>

// prototypes for sample functions to interpolate
/* complex single precision */
__host__ __device__ void ctestfun(const hipComplex *x, hipComplex *y);
__global__ void ctestkernel(int n, const hipComplex *in, int incin, hipComplex* out, int incout);
cuchebStatus_t ctestcaller(int n, const hipComplex *in, int incin, hipComplex* out, int incout);

// driver
int main(){

	// compute variables
	/* complex single */
	int cn = pow(2,3)+1;
	int sizeC = sizeof(hipComplex);
	hipComplex *cpts, *ca, *cb, *ccfs, *cfvs, *dcpts, *dca, *dcb, *dccfs, *dcfvs;
	
	// allocate host memory
	/* complex single */
	cuchebCheckError((void*)(cpts = (hipComplex*)malloc(cn*sizeC)),__FILE__,__LINE__);
	cuchebCheckError((void*)(ccfs = (hipComplex*)malloc(cn*sizeC)),__FILE__,__LINE__);
	cuchebCheckError((void*)(cfvs = (hipComplex*)malloc(cn*sizeC)),__FILE__,__LINE__);
	cuchebCheckError((void*)(ca = (hipComplex*)malloc(sizeC)),__FILE__,__LINE__);
	cuchebCheckError((void*)(cb = (hipComplex*)malloc(sizeC)),__FILE__,__LINE__);
	
	// allocate device memory
	/* complex single */
	cuchebCheckError(hipMalloc(&dcpts, cn*sizeC),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dccfs, cn*sizeC),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dcfvs, cn*sizeC),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dca, sizeC),__FILE__,__LINE__);
	cuchebCheckError(hipMalloc(&dcb, sizeC),__FILE__,__LINE__);
	
	// set host pointers
	/* complex single */
	*ca = make_hipFloatComplex(-1.0f,0.0f);
	*cb = make_hipFloatComplex(1.0f,0.0f);
	
	// copy host memory to device memory
	/* complex single */
	cuchebCheckError(hipMemcpy(dca, ca, sizeC, hipMemcpyHostToDevice),__FILE__,__LINE__);
	cuchebCheckError(hipMemcpy(dcb, cb, sizeC, hipMemcpyHostToDevice),__FILE__,__LINE__);
	
	// compute chebpoints
	/* complex single */
	cuchebCheckError(cuchebCpoints(cn,dca,dcb,dcpts,1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex single */
	cuchebCheckError(hipMemcpy(cpts, dcpts, cn*sizeC, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// compute funvals
	/* complex single */
	cuchebCheckError(ctestcaller(cn, dcpts, 1, dcfvs, 1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex single */
	cuchebCheckError(hipMemcpy(cfvs, dcfvs, cn*sizeC, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// compute chebcoeffs
	/* complex single */
	cuchebCheckError(cuchebCcoeffs(cn, dcfvs, 1, dccfs, 1),__FILE__,__LINE__);
	
	// copy device memory to host memory
	/* complex single */
	cuchebCheckError(hipMemcpy(ccfs, dccfs, cn*sizeC, hipMemcpyDeviceToHost),__FILE__,__LINE__);
	
	// print output
	/* complex single */
	printf("complex single precision\n");
	for(int ii=0;ii<cn;ii++){printf("cpts[%d] = (%+e,%+e), cfvs[%d] = (%+e,%+e), ccfs[%d] = (%+e,%+e)\n",
		ii,hipCrealf(cpts[ii]),hipCimagf(cpts[ii]),ii,hipCrealf(cfvs[ii]),hipCimagf(cfvs[ii]),ii,hipCrealf(ccfs[ii]),hipCimagf(ccfs[ii]));}
	printf("\n");

	// free device memory
	/* complex single */
	cuchebCheckError(hipFree(dcpts),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dccfs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dcfvs),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dca),__FILE__,__LINE__);
	cuchebCheckError(hipFree(dcb),__FILE__,__LINE__);

	// free host memory
	/* complex single */
	free(cpts);
	free(ccfs);
	free(cfvs);
	free(ca);
	free(cb);
	
	return 0;
}

// sample functions to interpolate
/* complex single precision */
__host__ __device__ void ctestfun(const hipComplex *x, hipComplex *y){
	hipComplex temp;
	hipComplex I = make_hipFloatComplex(1.0f,1.0f);
	temp = hipCmulf(*x,I);
	*y = hipCmulf(hipCmulf(temp,temp),temp);
}
__global__ void ctestkernel(int n, const hipComplex *in, int incin, hipComplex* out, int incout){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;

	if(ii < n){
		ctestfun(&in[ii*incin],&out[ii*incout]);
	}
}
cuchebStatus_t ctestcaller(int n, const hipComplex *in, int incin, hipComplex* out, int incout){
	// check n
	if(n <= 0){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check incin
	if(incin <= 0){
		fprintf(stderr,"\nIn %s line: %d, incin must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check incout
	if(incout <= 0){
		fprintf(stderr,"\nIn %s line: %d, incout must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
		
	// query device
	int dev;
	hipDeviceProp_t prop;
	cuchebCheckError(hipGetDevice(&dev),__FILE__,__LINE__);
	cuchebCheckError(hipGetDeviceProperties(&prop,dev),__FILE__,__LINE__);
	
	// set blockSize
	int blockSize;
	blockSize = prop.maxThreadsPerBlock;
	
	// set gridSize
	int gridSize;
	gridSize = (int)ceil((double)n/blockSize);
	
	// launch fill input kernel
	ctestkernel<<<gridSize,blockSize>>>(n, in, incin, out, incout);
	
	// check for kernel error
	cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
	// return success
	return CUCHEB_STATUS_SUCCESS;
}
