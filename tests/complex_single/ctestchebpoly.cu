#include "hip/hip_runtime.h"
#include <cucheb.h>

/* helpers for hipComplex precision constructors */
__global__ void cfunkernel(int n, const hipComplex *in, int incin, hipComplex *out, int incout);
cuchebStatus_t cfuncaller(int n, const hipComplex *in, int incin, hipComplex *out, int incout);

/* drvier */
int main(){

	// compute variables
	int deg;
	hipComplex a, b;
	float tol;
	
	// ChebPoly 1
	ChebPoly CP(CUCHEB_FIELD_FLOAT_COMPLEX);
	CP.printlong();
	
	// ChebPoly 2
	a = make_hipFloatComplex(-1.0f,0.0f);
	b = make_hipFloatComplex(1.0f,0.0f);
	deg = pow(2,3);
	ChebPoly CP2(&cfuncaller,&a,&b,deg);
	CP2.printlong();
	
	// ChebPoly 2
	a = make_hipFloatComplex(-1.0f,0.0f);
	b = make_hipFloatComplex(1.0f,0.0f);
	tol = 1e-5;
	ChebPoly CP3(&cfuncaller,&a,&b,&tol);
	CP3.print();
	
	// ChebPoly 4
	a = make_hipFloatComplex(-1.0f,0.0f);
	b = make_hipFloatComplex(1.0f,0.0f);
	ChebPoly CP4(&cfuncaller,&a,&b);
	CP4.print();

	// return	
	return 0;
}

/* helpers for hipComplex precision constructors */
/* kernel to call device function */
__global__ void cfunkernel(int n, const hipComplex *in, int incin, hipComplex *out, int incout){
	int tix = threadIdx.x, bix = blockIdx.x, bdx = blockDim.x;
	int ii = bix*bdx+tix;

	if(ii < n){
		out[ii*incout] = make_hipFloatComplex(123.0f*hipCabsf(in[ii*incin]),0.0f);
	}
}
/* subroutine to call dfunkernel */
cuchebStatus_t cfuncaller(int n, const hipComplex *in, int incin, hipComplex *out, int incout){
	
	// check n
	if(n <= 0){
		fprintf(stderr,"\nIn %s line: %d, n must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check incin
	if(incin <= 0){
		fprintf(stderr,"\nIn %s line: %d, incin must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// check incout
	if(incout <= 0){
		fprintf(stderr,"\nIn %s line: %d, incout must be > 0.\n",__FILE__,__LINE__);
		cuchebExit(-1);
	}
	
	// query device
	int dev;
	hipDeviceProp_t prop;
	cuchebCheckError(hipGetDevice(&dev),__FILE__,__LINE__);
	cuchebCheckError(hipGetDeviceProperties(&prop,dev),__FILE__,__LINE__);
	
	// set blockSize
	int blockSize;
	blockSize = prop.maxThreadsPerBlock;
	
	// set gridSize
	int gridSize;
	gridSize = (int)ceil((double)n/blockSize);
	
	// launch fill input kernel
	cfunkernel<<<gridSize,blockSize>>>(n, in, incin, out, incout);
	
	// check for kernel error
	cuchebCheckError(hipPeekAtLastError(),__FILE__,__LINE__);
	
	// return success
	return CUCHEB_STATUS_SUCCESS;
}
